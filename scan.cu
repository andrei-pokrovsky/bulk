#include "hip/hip_runtime.h"
#include <moderngpu.cuh>
#include <thrust/scan.h>
#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <cassert>
#include <iostream>
#include "time_invocation_cuda.hpp"
#include <thrust/detail/temporary_array.h>
#include <thrust/copy.h>
#include <bulk/bulk.hpp>


typedef int T;


template<unsigned int size, unsigned int grainsize>
struct inclusive_scan_n
{
  template<typename InputIterator, typename Size, typename OutputIterator, typename BinaryFunction>
  __device__ void operator()(bulk::static_thread_group<size,grainsize> &this_group, InputIterator first, Size n, OutputIterator result, BinaryFunction binary_op)
  {
    bulk::inclusive_scan(this_group, first, first + n, result, binary_op);
  }
};


template<unsigned int size, unsigned int grainsize>
struct exclusive_scan_n
{
  template<typename InputIterator, typename Size, typename OutputIterator, typename T, typename BinaryFunction>
  __device__ void operator()(bulk::static_thread_group<size,grainsize> &this_group, InputIterator first, Size n, OutputIterator result, T init, BinaryFunction binary_op)
  {
    bulk::exclusive_scan(this_group, first, first + n, result, init, binary_op);
  }
};


template<unsigned int size, typename ThreadGroup, typename T, typename BinaryFunction>
__device__ T small_inplace_exclusive_scan_with_buffer(ThreadGroup &g, T *first, T init, T *buffer, BinaryFunction binary_op)
{
  // XXX int is noticeably faster than ThreadGroup::size_type
  typedef int size_type;
  //typedef typename ThreadGroup::size_type size_type;

  // ping points to the most current data
  T *ping = first;
  T *pong = buffer;

  size_type tid = g.this_thread.index();

  if(tid == 0)
  {
    first[0] = binary_op(init, first[0]);
  }

  T x = first[tid];

  g.wait();

  #pragma unroll
  for(size_type offset = 1; offset < size; offset += offset)
  {
    if(tid >= offset)
    {
      x = binary_op(ping[tid - offset], x);
    }

    thrust::swap(ping, pong);

    ping[tid] = x;

    g.wait();
  }

  T result = ping[size - 1];

  x = (tid == 0) ? init : ping[tid - 1];

  g.wait();

  first[tid] = x;

  g.wait();

  return result;
}


template<typename Tuning, typename InputIt, typename OutputIt, typename T, typename BinaryFunction>
__global__ void inclusive_scan_kernel(InputIt data_global, int count, int2 task, const T* reduction_global, OutputIt dest_global, BinaryFunction binary_op)
{
  typedef MGPU_LAUNCH_PARAMS Params;
  const int groupsize = Params::NT;
  const int grainsize = Params::VT;
  const int elements_per_group = groupsize * grainsize;

  bulk::static_thread_group<groupsize,grainsize> this_group;

  typedef typename thrust::iterator_value<InputIt>::type  input_type;
  // XXX this needs to be inferred from the iterators and binary op
  typedef typename thrust::iterator_value<OutputIt>::type intermediate_type;
  
  typedef mgpu::CTAScan<groupsize, mgpu::ScanOp<mgpu::ScanOpTypeAdd,int> > S;

  union Shared {
    input_type          inputs[elements_per_group];
    intermediate_type   results[elements_per_group];
  };
  __shared__ Shared shared;

  __shared__ intermediate_type s_sums[groupsize];
  __shared__ intermediate_type s_scan_buffer[groupsize];
  
  int tid = threadIdx.x;
  int block = blockIdx.x;
  int2 range = mgpu::ComputeTaskRange(block, task, elements_per_group, count);
  
  // give block 0 a carry by taking the first input element
  // and adjusting its range
  T carry = (block != 0) ? reduction_global[block] : data_global[0];
  if(block == 0)
  {
    if(tid == 0)
    {
      *dest_global = carry;
    }

    ++range.x;
  }

  for(; range.x < range.y; range.x += elements_per_group)
  {
    int partition_size = thrust::min<int>(elements_per_group, range.y - range.x);
    
    // stage data through shared memory
    bulk::copy_n(this_group, data_global + range.x, partition_size, shared.inputs);
    
    // Transpose out of shared memory.
    input_type local_inputs[grainsize];

    int local_offset = grainsize * tid;

    int local_size = thrust::max<int>(0,thrust::min<int>(grainsize, partition_size - grainsize * tid));

    // XXX this should be uninitialized<input_type>
    input_type x;

    // this loop is a fused copy and accumulate
    #pragma unroll
    for(int i = 0; i < grainsize; ++i)
    {
      int index = local_offset + i;
      if(index < partition_size)
      {
        local_inputs[i] = shared.inputs[index];
        x = i ? binary_op(x, local_inputs[i]) : local_inputs[i];
      }
    }
    if(local_size)
    {
      s_sums[tid] = x;
    }
    this_group.wait();
    
    carry = small_inplace_exclusive_scan_with_buffer<groupsize>(this_group, s_sums, carry, s_scan_buffer, binary_op);

    if(local_size)
    {
      x = s_sums[tid];
    }
    
    // this loop is an inclusive_scan
    // XXX this loop should be one of the things to modify when porting to exclusive_scan
    #pragma unroll
    for(int i = 0; i < grainsize; ++i) 
    {
      int index = local_offset + i;
      if(index < partition_size)
      {
        x = binary_op(x, local_inputs[i]);

        shared.results[index] = x;
      }
    }
    this_group.wait();
    
    bulk::copy_n(this_group, shared.results, partition_size, dest_global + range.x);
  }
}


template<mgpu::MgpuScanType Type, typename InputIt, typename OutputIt, typename Op>
void IncScan(InputIt data_global, int count, OutputIt dest_global, Op op, mgpu::CudaContext& context)
{
  typedef typename Op::value_type value_type;
  typedef typename Op::result_type result_type;
  
  const int threshold_of_parallelism = 20000;

  if(count < threshold_of_parallelism)
  {
    const int size = 512;
    const int grainsize = 3;

    bulk::static_thread_group<size,grainsize> group;
    bulk::async(bulk::par(group, 1), inclusive_scan_n<size,grainsize>(), bulk::there, data_global, count, dest_global, thrust::plus<int>());
  }
  else
  {
    // Run the parallel raking reduce as an upsweep.
    const int NT = 128;
    const int VT = 7;
    typedef mgpu::LaunchBoxVT<NT, VT> Tuning;
    int2 launch = Tuning::GetLaunchParams(context);
    const int NV = launch.x * launch.y;
    
    int numTiles = MGPU_DIV_UP(count, NV);
    int numBlocks = std::min(context.NumSMs() * 25, numTiles);
    int2 task = mgpu::DivideTaskRange(numTiles, numBlocks);
    
    MGPU_MEM(value_type) reductionDevice = context.Malloc<value_type>(numBlocks + 1);
    	
    mgpu::KernelReduce<Tuning><<<numBlocks, launch.x>>>(data_global, count, task, reductionDevice->get(), op);
    
    // Run a parallel latency-oriented scan to reduce the spine of the 
    // raking reduction.
    const unsigned int groupsize2 = 256;
    const unsigned int grainsize2 = 3;

    bulk::static_thread_group<groupsize2,grainsize2> group;
    bulk::async(bulk::par(group,1), exclusive_scan_n<groupsize2,grainsize2>(), bulk::there, reductionDevice->get(), numBlocks, reductionDevice->get(), 0, thrust::plus<int>());
    
    // Run a raking scan as a downsweep.
    inclusive_scan_kernel<Tuning><<<numBlocks, launch.x>>>(data_global, count, task, reductionDevice->get(), dest_global, thrust::plus<int>());
  }
}


template<typename InputIterator, typename OutputIterator>
OutputIterator my_inclusive_scan(InputIterator first, InputIterator last, OutputIterator result)
{
  mgpu::ContextPtr ctx = mgpu::CreateCudaDevice(0);

  ::IncScan<mgpu::MgpuScanTypeInc>(thrust::raw_pointer_cast(&*first),
                                   last - first,
                                   thrust::raw_pointer_cast(&*result),
                                   mgpu::ScanOp<mgpu::ScanOpTypeAdd,int>(),
                                   *ctx);

  return result + (last - first);
}


void my_scan(thrust::device_vector<T> *data)
{
  my_inclusive_scan(data->begin(), data->end(), data->begin());
}


void do_it(size_t n)
{
  thrust::host_vector<T> h_input(n);
  thrust::fill(h_input.begin(), h_input.end(), 1);

  thrust::host_vector<T> h_result(n);

  thrust::inclusive_scan(h_input.begin(), h_input.end(), h_result.begin());

  thrust::device_vector<T> d_input = h_input;
  thrust::device_vector<T> d_result(d_input.size());

  my_inclusive_scan(d_input.begin(), d_input.end(), d_result.begin());

  hipError_t error = hipDeviceSynchronize();

  if(error)
  {
    std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
  }

  assert(h_result == d_result);
}


template<typename InputIterator, typename OutputIterator>
OutputIterator mgpu_inclusive_scan(InputIterator first, InputIterator last, OutputIterator result)
{
  mgpu::ContextPtr ctx = mgpu::CreateCudaDevice(0);

  mgpu::Scan<mgpu::MgpuScanTypeInc>(thrust::raw_pointer_cast(&*first),
                                    last - first,
                                    thrust::raw_pointer_cast(&*result),
                                    mgpu::ScanOp<mgpu::ScanOpTypeAdd,int>(),
                                    (int*)0,
                                    false,
                                    *ctx);

  return result + (last - first);
}


void sean_scan(thrust::device_vector<T> *data)
{
  mgpu_inclusive_scan(data->begin(), data->end(), data->begin());
}


int main()
{
  for(size_t n = 1; n <= 1 << 20; n <<= 1)
  {
    std::cout << "Testing n = " << n << std::endl;
    do_it(n);
  }

  thrust::default_random_engine rng;
  for(int i = 0; i < 20; ++i)
  {
    size_t n = rng() % (1 << 20);
   
    std::cout << "Testing n = " << n << std::endl;
    do_it(n);
  }

  thrust::device_vector<T> vec(1 << 28);

  sean_scan(&vec);
  double sean_msecs = time_invocation_cuda(50, sean_scan, &vec);

  my_scan(&vec);
  double my_msecs = time_invocation_cuda(50, my_scan, &vec);

  std::cout << "Sean's time: " << sean_msecs << " ms" << std::endl;
  std::cout << "My time: " << my_msecs << " ms" << std::endl;

  std::cout << "My relative performance: " << sean_msecs / my_msecs << std::endl;

  return 0;
}

