#include "hip/hip_runtime.h"
#include <moderngpu.cuh>
#include <thrust/scan.h>
#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <cassert>
#include <iostream>
#include "time_invocation_cuda.hpp"
#include <thrust/detail/temporary_array.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include <bulk/bulk.hpp>


typedef int T;


template<unsigned int grainsize, typename Iterator1, typename Size, typename Iterator2>
__device__
void copy_n_with_grainsize(Iterator1 first, Size n, Iterator2 result)
{
  for(Iterator1 last = first + n;
      first < last;
      first += grainsize, result += grainsize)
  {
    for(int i = 0; i < grainsize; ++i)
    {
      if(i < (last - first))
      {
        result[i] = first[i];
      }
    }
  }
}


template<typename ThreadGroup, typename Iterator, typename Size, typename T, typename BinaryFunction>
__device__ T exclusive_scan_n(ThreadGroup &g, Iterator first, Size n, T init, T *carry_out, BinaryFunction binary_op)
{
  T x;

  int tid = g.this_thread.index();

  if(n > 0 && tid == 0)
  {
    *first = binary_op(init, *first);
  }

  if(tid < n)
  {
    x = first[tid];
  }

  g.wait();

  for(int offset = 1; offset < n; offset += offset)
  {
    if(tid >= offset && tid < n)
    {
      x = binary_op(first[tid - offset], x);
    }

    g.wait();

    if(tid < n)
    {
      first[tid] = x;
    }

    g.wait();
  }

  *carry_out = n > 0 ? first[n - 1] : init;

  if(tid - 1 < n)
  {
    x = tid ? first[tid - 1] : init;
  }

  g.wait();
  
  return x;
}


template<typename ThreadGroup, typename Iterator, typename Size, typename BinaryFunction>
__device__ void inclusive_scan_n(ThreadGroup &g, Iterator first, Size n, BinaryFunction binary_op)
{
  T x;

  int tid = g.this_thread.index();

  if(tid < n)
  {
    x = first[tid];
  }

  g.wait();

  for(int offset = 1; offset < n; offset += offset)
  {
    if(tid >= offset && tid < n)
    {
      x = binary_op(first[tid - offset], x);
    }

    g.wait();

    if(tid < n)
    {
      first[tid] = x;
    }

    g.wait();
  }
}


template<typename ThreadGroup, typename Iterator, typename Size, typename T, typename BinaryFunction>
__device__ T exclusive_scan_n(ThreadGroup &g, Iterator first, Size n, T init, BinaryFunction binary_op)
{
  T x;

  int tid = g.this_thread.index();

  if(n > 0 && tid == 0)
  {
    *first = binary_op(init, *first);
  }

  g.wait();

  if(tid < n)
  {
    x = first[tid];
  }

  g.wait();

  inclusive_scan_n(g, first, n, binary_op);

  T result = n > 0 ? first[n - 1] : init;

  x = (tid == 0 || tid - 1 >= n) ? init : first[tid - 1];

  g.wait();

  if(tid < n)
  {
    first[tid] = x;
  }

  g.wait();

  return result;
}


template<unsigned int size, unsigned int grainsize, bool inclusive>
struct scan
{
  template<typename InputIterator, typename OutputIterator, typename BinaryFunction>
  __device__ void operator()(bulk::static_thread_group<size,grainsize> &this_group, InputIterator cta_global, int count, OutputIterator dest_global, BinaryFunction binary_op)
  {
    typedef typename thrust::iterator_value<InputIterator>::type input_type;

    // XXX this needs to be inferred from the iterators and binary_op
    typedef typename thrust::iterator_value<OutputIterator>::type intermediate_type;
  
    const unsigned int elements_per_group = size * grainsize;

    // we don't need the inputs and the results at the same time
    // so we can overlay these arrays
    union stage
    {
      input_type *inputs;
      intermediate_type *results;
    };

    stage s_stage;
    s_stage.inputs = reinterpret_cast<input_type*>(bulk::malloc(this_group, elements_per_group * thrust::max<int>(sizeof(input_type), sizeof(intermediate_type))));

    intermediate_type *s_sums = reinterpret_cast<intermediate_type*>(bulk::malloc(this_group, size * sizeof(intermediate_type)));
    
    unsigned int tid = this_group.this_thread.index();
    
    // carry is the sum over all previous iterations
    intermediate_type carry = cta_global[0];
  
    if(this_group.this_thread.index() == 0)
    {
      dest_global[0] = carry;
    }
  
    for(int start = 1; start < count; start += elements_per_group)
    {
      int count2 = min(elements_per_group, count - start);
  
      // stage data through shared memory
      bulk::copy_n(this_group, cta_global + start, count2, s_stage.inputs);
      
      // Transpose data into register in thread order. Reduce terms serially.
      input_type local_inputs[grainsize];
  
      int local_size = max(0,min(grainsize, count2 - grainsize * tid));
  
      int local_offset = grainsize * tid;
  
      intermediate_type x = 0;
  
      if(local_size > 0)
      {
        // XXX would be cool simply to call
        // bulk::copy_n(this_group.this_thread, ...) instead
        copy_n_with_grainsize<grainsize>(s_stage.inputs + local_offset, local_size, local_inputs);
  
        // XXX this should actually be accumulate because we desire non-commutativity
        x = thrust::reduce(thrust::seq, local_inputs + 1, local_inputs + local_size, local_inputs[0], binary_op);
  
        s_sums[tid] = x;
      }
  
      this_group.wait();
  
      // scan this group's sums
      // XXX is this really the correct number of sums?
      //     it should be divide_ri(count2, grainsize)
      carry = ::exclusive_scan_n(this_group, s_sums, min(size,count2), carry, binary_op);
  
      // each thread does an inplace scan locally while incorporating the carries
      if(local_size > 0)
      {
        x = s_sums[tid];
  
        if(inclusive)
        {
          local_inputs[0] = binary_op(x,local_inputs[0]);
  
          // XXX would be cool simply to call
          // bulk::inclusive_scan(this_group.this_thread, ...) instead
          thrust::inclusive_scan(thrust::seq, local_inputs, local_inputs + local_size, local_inputs, binary_op);
        }
        else
        {
          // XXX would be cool simply to call
          // bulk::exclusive_scan(this_group.this_thread, ...) instead
          thrust::exclusive_scan(thrust::seq, local_inputs, local_inputs + local_size, local_inputs, x, binary_op);
        }
  
        // XXX would be cool simply to call
        // bulk::copy_n(this_group.this_thread, ...) instead
        copy_n_with_grainsize<grainsize>(local_inputs, local_size, s_stage.results + local_offset);
      }
  
      this_group.wait();
      
      // store results
      bulk::copy_n(this_group, s_stage.results, count2, dest_global + start);
    }

    bulk::free(this_group, s_stage.inputs);
    bulk::free(this_group, s_sums);
  }
};


template<mgpu::MgpuScanType Type, typename InputIt, typename OutputIt, typename Op>
void IncScan(InputIt data_global, int count, OutputIt dest_global, Op op, mgpu::CudaContext& context)
{
  typedef typename Op::value_type value_type;
  typedef typename Op::result_type result_type;
  
  const int CutOff = 20000;

  if(count < CutOff)
  {
    const int NT = 512;
    const int VT = 3;

    bulk::static_thread_group<NT,VT> group;
    bulk::async(bulk::par(group, 1), scan<NT,VT,true>(), bulk::there, data_global, count, dest_global, thrust::plus<int>());
  }
  else
  {
    // Run the parallel raking reduce as an upsweep.
    const int NT = 128;
    const int VT = 7;
    typedef mgpu::LaunchBoxVT<NT, VT> Tuning;
    int2 launch = Tuning::GetLaunchParams(context);
    const int NV = launch.x * launch.y;
    
    int numTiles = MGPU_DIV_UP(count, NV);
    int numBlocks = std::min(context.NumSMs() * 25, numTiles);
    int2 task = mgpu::DivideTaskRange(numTiles, numBlocks);
    
    MGPU_MEM(value_type) reductionDevice = context.Malloc<value_type>(numBlocks + 1);
    	
    mgpu::KernelReduce<Tuning><<<numBlocks, launch.x>>>(data_global, count, task, reductionDevice->get(), op);
    
    // Run a parallel latency-oriented scan to reduce the spine of the 
    // raking reduction.
    const int NT2 = 256;
    const int VT2 = 3;

    bulk::static_thread_group<NT2,VT2> group;
    bulk::async(bulk::par(group,1), scan<NT2,VT2,false>(), bulk::there, reductionDevice->get(), numBlocks, reductionDevice->get(), thrust::plus<int>());
    
    // Run a raking scan as a downsweep.
    mgpu::KernelScanDownsweep<Tuning, Type><<<numBlocks, launch.x>>>(data_global, count, task, reductionDevice->get(), dest_global, false, op);
  }
}


template<typename InputIterator, typename OutputIterator>
OutputIterator my_inclusive_scan(InputIterator first, InputIterator last, OutputIterator result)
{
  mgpu::ContextPtr ctx = mgpu::CreateCudaDevice(0);

  ::IncScan<mgpu::MgpuScanTypeInc>(thrust::raw_pointer_cast(&*first),
                                   last - first,
                                   thrust::raw_pointer_cast(&*result),
                                   mgpu::ScanOp<mgpu::ScanOpTypeAdd,int>(),
                                   *ctx);

  return result + (last - first);
}


void my_scan(thrust::device_vector<T> *data)
{
  my_inclusive_scan(data->begin(), data->end(), data->begin());
}


void do_it(size_t n)
{
  thrust::host_vector<T> h_input(n);
  thrust::fill(h_input.begin(), h_input.end(), 1);

  thrust::host_vector<T> h_result(n);

  thrust::inclusive_scan(h_input.begin(), h_input.end(), h_result.begin());

  thrust::device_vector<T> d_input = h_input;
  thrust::device_vector<T> d_result(d_input.size());

  my_inclusive_scan(d_input.begin(), d_input.end(), d_result.begin());

  hipError_t error = hipDeviceSynchronize();

  if(error)
  {
    std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
  }

  assert(h_result == d_result);
}


template<typename InputIterator, typename OutputIterator>
OutputIterator mgpu_inclusive_scan(InputIterator first, InputIterator last, OutputIterator result)
{
  mgpu::ContextPtr ctx = mgpu::CreateCudaDevice(0);

  mgpu::Scan<mgpu::MgpuScanTypeInc>(thrust::raw_pointer_cast(&*first),
                                    last - first,
                                    thrust::raw_pointer_cast(&*result),
                                    mgpu::ScanOp<mgpu::ScanOpTypeAdd,int>(),
                                    (int*)0,
                                    false,
                                    *ctx);

  return result + (last - first);
}


void sean_scan(thrust::device_vector<T> *data)
{
  mgpu_inclusive_scan(data->begin(), data->end(), data->begin());
}


int main()
{
  for(size_t n = 1; n <= 1 << 20; n <<= 1)
  {
    std::cout << "Testing n = " << n << std::endl;
    do_it(n);
  }

  thrust::default_random_engine rng;
  for(int i = 0; i < 20; ++i)
  {
    size_t n = rng() % (1 << 20);
   
    std::cout << "Testing n = " << n << std::endl;
    do_it(n);
  }

  thrust::device_vector<T> vec(1 << 24);

  sean_scan(&vec);
  double sean_msecs = time_invocation_cuda(50, sean_scan, &vec);

  my_scan(&vec);
  double my_msecs = time_invocation_cuda(50, my_scan, &vec);

  std::cout << "Sean's time: " << sean_msecs << " ms" << std::endl;
  std::cout << "My time: " << my_msecs << " ms" << std::endl;

  std::cout << "My relative performance: " << sean_msecs / my_msecs << std::endl;

  return 0;
}

