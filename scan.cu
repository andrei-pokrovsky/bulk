#include "hip/hip_runtime.h"
#include <moderngpu.cuh>
#include <thrust/scan.h>
#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <cassert>
#include <iostream>
#include "time_invocation_cuda.hpp"
#include <thrust/detail/temporary_array.h>
#include <thrust/copy.h>
#include <bulk/bulk.hpp>


typedef int T;


template<unsigned int size, unsigned int grainsize>
struct inclusive_scan_n
{
  template<typename InputIterator, typename Size, typename OutputIterator, typename BinaryFunction>
  __device__ void operator()(bulk::static_thread_group<size,grainsize> &this_group, InputIterator first, Size n, OutputIterator result, BinaryFunction binary_op)
  {
    bulk::inclusive_scan(this_group, first, first + n, result, binary_op);
  }
};


template<unsigned int size, unsigned int grainsize>
struct exclusive_scan_n
{
  template<typename InputIterator, typename Size, typename OutputIterator, typename T, typename BinaryFunction>
  __device__ void operator()(bulk::static_thread_group<size,grainsize> &this_group, InputIterator first, Size n, OutputIterator result, T init, BinaryFunction binary_op)
  {
    bulk::exclusive_scan(this_group, first, first + n, result, init, binary_op);
  }
};


template<unsigned int size, typename ThreadGroup, typename T, typename BinaryFunction>
__device__ T small_inplace_exclusive_scan_with_buffer(ThreadGroup &g, T *first, T init, T *buffer, BinaryFunction binary_op)
{
  // XXX int is noticeably faster than ThreadGroup::size_type
  typedef int size_type;
  //typedef typename ThreadGroup::size_type size_type;

  // ping points to the most current data
  T *ping = first;
  T *pong = buffer;

  size_type tid = g.this_thread.index();

  if(tid == 0)
  {
    first[0] = binary_op(init, first[0]);
  }

  T x = first[tid];

  g.wait();

  #pragma unroll
  for(size_type offset = 1; offset < size; offset += offset)
  {
    if(tid >= offset)
    {
      x = binary_op(ping[tid - offset], x);
    }

    thrust::swap(ping, pong);

    ping[tid] = x;

    g.wait();
  }

  T result = ping[size - 1];

  x = (tid == 0) ? init : ping[tid - 1];

  g.wait();

  first[tid] = x;

  g.wait();

  return result;
}


template<std::size_t groupsize, std::size_t grainsize, typename RandomAccessIterator1, typename RandomAccessIterator2, typename T, typename BinaryFunction>
__device__ void inclusive_scan_with_carry_with_buffer(bulk::static_thread_group<groupsize,grainsize> &g,
                                                      RandomAccessIterator1 first, RandomAccessIterator1 last,
                                                      RandomAccessIterator2 result,
                                                      T carry_in,
                                                      BinaryFunction binary_op,
                                                      void *buffer)
{
  typedef typename thrust::iterator_value<RandomAccessIterator1>::type  input_type;
  // XXX this needs to be inferred from the iterators and binary op
  typedef typename thrust::iterator_value<RandomAccessIterator2>::type intermediate_type;

  intermediate_type *s_sums = reinterpret_cast<intermediate_type*>(buffer);

  union {
    input_type        *inputs;
    intermediate_type *results;
  } shared;

  shared.inputs = reinterpret_cast<intermediate_type*>(reinterpret_cast<char*>(buffer) + 2*groupsize*sizeof(intermediate_type));

  // XXX int is noticeably faster than ThreadGroup::size_type
  //typedef typename bulk::static_thread_group<groupsize,grainsize>::size_type size_type;
  typedef int size_type;

  size_type tid = g.this_thread.index();

  size_type elements_per_group = groupsize * grainsize;

  for(; first < last; first += elements_per_group, result += elements_per_group)
  {
    size_type partition_size = thrust::min<size_type>(elements_per_group, last - first);
    
    // stage data through shared memory
    bulk::copy_n(g, first, partition_size, shared.inputs);
    
    // Transpose out of shared memory.
    input_type local_inputs[grainsize];

    size_type local_offset = grainsize * tid;

    size_type local_size = thrust::max<size_type>(0,thrust::min<size_type>(grainsize, partition_size - grainsize * tid));

    // XXX this should be uninitialized<input_type>
    input_type x;

    // this loop is a fused copy and accumulate
    #pragma unroll
    for(size_type i = 0; i < grainsize; ++i)
    {
      size_type index = local_offset + i;
      if(index < partition_size)
      {
        local_inputs[i] = shared.inputs[index];
        x = i ? binary_op(x, local_inputs[i]) : local_inputs[i];
      } // end if
    } // end for

    if(local_size)
    {
      s_sums[tid] = x;
    } // end if

    g.wait();
    
    // exclusive scan the array of per-thread sums
    carry_in = small_inplace_exclusive_scan_with_buffer<groupsize>(g, s_sums, carry_in, s_sums + groupsize, binary_op);

    if(local_size)
    {
      x = s_sums[tid];
    } // end if
    
    // this loop is an inclusive_scan_with_carry (x begins as the carry)
    // XXX this loop should be one of the things to modify when porting to exclusive_scan
    #pragma unroll
    for(size_type i = 0; i < grainsize; ++i) 
    {
      size_type index = local_offset + i;
      if(index < partition_size)
      {
        x = binary_op(x, local_inputs[i]);

        shared.results[index] = x;
      } // end if
    } // end for

    g.wait();
    
    bulk::copy_n(g, shared.results, partition_size, result);
  } // end for
}


template<std::size_t groupsize, std::size_t grainsize, typename RandomAccessIterator1, typename RandomAccessIterator2, typename T, typename BinaryFunction>
__device__ void inclusive_scan_with_carry(bulk::static_thread_group<groupsize,grainsize> &g,
                                          RandomAccessIterator1 first, RandomAccessIterator1 last,
                                          RandomAccessIterator2 result,
                                          T carry_in,
                                          BinaryFunction binary_op)
{
  typedef typename thrust::iterator_value<RandomAccessIterator1>::type  input_type;
  // XXX this needs to be inferred from the iterators and binary op
  typedef typename thrust::iterator_value<RandomAccessIterator2>::type intermediate_type;

  int num_stage_bytes = groupsize * grainsize * thrust::max<int>(sizeof(input_type),sizeof(intermediate_type));
  int num_sums_bytes = 2 * groupsize * sizeof(intermediate_type);

  void *buffer = bulk::malloc(g, num_stage_bytes + num_sums_bytes);

  if(bulk::detail::is_shared(buffer))
  {
    inclusive_scan_with_carry_with_buffer(g, first, last, result, carry_in, binary_op, bulk::detail::on_chip_cast(buffer));
  }
  else
  {
    inclusive_scan_with_carry_with_buffer(g, first, last, result, carry_in, binary_op, buffer);
  }

  bulk::free(g, buffer);
} // end inclusive_scan_with_carry()


template<std::size_t groupsize, std::size_t grainsize>
struct inclusive_downsweep
{
  template<typename RandomAccessIterator1, typename RandomAccessIterator2, typename T, typename BinaryFunction>
  __device__ void operator()(bulk::static_thread_group<groupsize,grainsize> &this_group,
                             RandomAccessIterator1 first,
                             int count,
                             int2 task,
                             const T *carries,
                             RandomAccessIterator2 result,
                             BinaryFunction binary_op)
  {
    const int elements_per_group = groupsize * grainsize;
  
    int2 range = mgpu::ComputeTaskRange(this_group.index(), task, elements_per_group, count);
    
    // give group 0 a carry by taking the first input element
    // and adjusting its range
    T carry = (this_group.index() != 0) ? carries[this_group.index()] : first[0];
    if(this_group.index() == 0)
    {
      if(this_group.this_thread.index() == 0)
      {
        *result = carry;
      }
  
      ++range.x;
    }
  
    RandomAccessIterator1 last = first + range.y;
    first += range.x;
    result += range.x;
  
    inclusive_scan_with_carry(this_group, first, last, result, carry, binary_op);
  }
};


template<mgpu::MgpuScanType Type, typename InputIt, typename OutputIt, typename Op>
void IncScan(InputIt data_global, int count, OutputIt dest_global, Op op, mgpu::CudaContext& context)
{
  typedef typename Op::value_type value_type;
  typedef typename Op::result_type result_type;
  
  const int threshold_of_parallelism = 20000;

  if(count < threshold_of_parallelism)
  {
    const int size = 512;
    const int grainsize = 3;

    bulk::static_thread_group<size,grainsize> group;
    bulk::async(bulk::par(group, 1), inclusive_scan_n<size,grainsize>(), bulk::there, data_global, count, dest_global, thrust::plus<int>());
  }
  else
  {
    // Run the parallel raking reduce as an upsweep.
    const int groupsize1 = 128;
    const int grainsize1 = 7;
    typedef mgpu::LaunchBoxVT<groupsize1, grainsize1> Tuning;
    int2 launch = Tuning::GetLaunchParams(context);
    const int NV = launch.x * launch.y;
    
    int numTiles = MGPU_DIV_UP(count, NV);
    int numBlocks = std::min(context.NumSMs() * 25, numTiles);
    int2 task = mgpu::DivideTaskRange(numTiles, numBlocks);
    
    MGPU_MEM(value_type) reductionDevice = context.Malloc<value_type>(numBlocks + 1);
    	
    // N loads
    mgpu::KernelReduce<Tuning><<<numBlocks, launch.x>>>(data_global, count, task, reductionDevice->get(), op);
    
    // scan the sums to get the carries
    const unsigned int groupsize2 = 256;
    const unsigned int grainsize2 = 3;

    // XXX we could scatter the carries to the output instead of scanning in place
    //     this might simplify the next kernel
    bulk::static_thread_group<groupsize2,grainsize2> group2;
    bulk::async(bulk::par(group2,1), exclusive_scan_n<groupsize2,grainsize2>(), bulk::there, reductionDevice->get(), numBlocks, reductionDevice->get(), 0, thrust::plus<int>());
    
    // do the downsweep - N loads, N stores
    bulk::static_thread_group<groupsize1,grainsize1> group1;
    bulk::async(bulk::par(group1,numBlocks), inclusive_downsweep<groupsize1,grainsize1>(), bulk::there, data_global, count, task, reductionDevice->get(), dest_global, thrust::plus<int>());
  }
}


template<typename InputIterator, typename OutputIterator>
OutputIterator my_inclusive_scan(InputIterator first, InputIterator last, OutputIterator result)
{
  mgpu::ContextPtr ctx = mgpu::CreateCudaDevice(0);

  ::IncScan<mgpu::MgpuScanTypeInc>(thrust::raw_pointer_cast(&*first),
                                   last - first,
                                   thrust::raw_pointer_cast(&*result),
                                   mgpu::ScanOp<mgpu::ScanOpTypeAdd,int>(),
                                   *ctx);

  return result + (last - first);
}


void my_scan(thrust::device_vector<T> *data)
{
  my_inclusive_scan(data->begin(), data->end(), data->begin());
}


void do_it(size_t n)
{
  thrust::host_vector<T> h_input(n);
  thrust::fill(h_input.begin(), h_input.end(), 1);

  thrust::host_vector<T> h_result(n);

  thrust::inclusive_scan(h_input.begin(), h_input.end(), h_result.begin());

  thrust::device_vector<T> d_input = h_input;
  thrust::device_vector<T> d_result(d_input.size());

  my_inclusive_scan(d_input.begin(), d_input.end(), d_result.begin());

  hipError_t error = hipDeviceSynchronize();

  if(error)
  {
    std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
  }

  assert(h_result == d_result);
}


template<typename InputIterator, typename OutputIterator>
OutputIterator mgpu_inclusive_scan(InputIterator first, InputIterator last, OutputIterator result)
{
  mgpu::ContextPtr ctx = mgpu::CreateCudaDevice(0);

  mgpu::Scan<mgpu::MgpuScanTypeInc>(thrust::raw_pointer_cast(&*first),
                                    last - first,
                                    thrust::raw_pointer_cast(&*result),
                                    mgpu::ScanOp<mgpu::ScanOpTypeAdd,int>(),
                                    (int*)0,
                                    false,
                                    *ctx);

  return result + (last - first);
}


void sean_scan(thrust::device_vector<T> *data)
{
  mgpu_inclusive_scan(data->begin(), data->end(), data->begin());
}


int main()
{
  for(size_t n = 1; n <= 1 << 20; n <<= 1)
  {
    std::cout << "Testing n = " << n << std::endl;
    do_it(n);
  }

  thrust::default_random_engine rng;
  for(int i = 0; i < 20; ++i)
  {
    size_t n = rng() % (1 << 20);
   
    std::cout << "Testing n = " << n << std::endl;
    do_it(n);
  }

  thrust::device_vector<T> vec(1 << 28);

  sean_scan(&vec);
  double sean_msecs = time_invocation_cuda(50, sean_scan, &vec);

  my_scan(&vec);
  double my_msecs = time_invocation_cuda(50, my_scan, &vec);

  std::cout << "Sean's time: " << sean_msecs << " ms" << std::endl;
  std::cout << "My time: " << my_msecs << " ms" << std::endl;

  std::cout << "My relative performance: " << sean_msecs / my_msecs << std::endl;

  return 0;
}

