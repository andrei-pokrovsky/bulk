#include "hip/hip_runtime.h"
#include <moderngpu.cuh>
#include <thrust/scan.h>
#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <cassert>
#include <iostream>
#include "time_invocation_cuda.hpp"
#include <thrust/detail/temporary_array.h>
#include <thrust/copy.h>
#include <bulk/bulk.hpp>


typedef int T;


template<unsigned int size, unsigned int grainsize>
struct inclusive_scan_n
{
  template<typename InputIterator, typename Size, typename OutputIterator, typename BinaryFunction>
  __device__ void operator()(bulk::static_thread_group<size,grainsize> &this_group, InputIterator first, Size n, OutputIterator result, BinaryFunction binary_op)
  {
    bulk::inclusive_scan(this_group, first, first + n, result, binary_op);
  }
};


template<std::size_t groupsize, std::size_t grainsize>
struct inclusive_downsweep
{
  template<typename RandomAccessIterator1, typename RandomAccessIterator2, typename T, typename BinaryFunction>
  __device__ void operator()(bulk::static_thread_group<groupsize,grainsize> &this_group,
                             RandomAccessIterator1 first,
                             int count,
                             int2 task,
                             const T *carries,
                             RandomAccessIterator2 result,
                             BinaryFunction binary_op)
  {
    const int elements_per_group = groupsize * grainsize;
  
    int2 range = mgpu::ComputeTaskRange(this_group.index(), task, elements_per_group, count);
    
    // give group 0 a carry by taking the first input element
    // and adjusting its range
    T carry = (this_group.index() != 0) ? carries[this_group.index()-1] : first[0];
    if(this_group.index() == 0)
    {
      if(this_group.this_thread.index() == 0)
      {
        *result = carry;
      }
  
      ++range.x;
    }
  
    RandomAccessIterator1 last = first + range.y;
    first += range.x;
    result += range.x;
  
    bulk::detail::scan_detail::inclusive_scan_with_carry(this_group, first, last, result, carry, binary_op);
  }
};


template<typename Tuning, typename InputIt, typename Op>
__global__ void my_KernelReduce(InputIt data_global, int count, int2 task, typename Op::value_type* reduction_global, Op op)
{
  typedef MGPU_LAUNCH_PARAMS Params;
  const int groupsize = Params::NT;
  const int grainsize = Params::VT;
  const int elements_per_group = groupsize * grainsize;
  typedef typename Op::input_type input_type;
  typedef typename Op::value_type value_type;
  typedef mgpu::CTAReduce<groupsize, Op> R;
  
  union Shared
  {
    typename R::Storage reduce;
    input_type inputs[elements_per_group];
  };
  __shared__ Shared shared;
  
  int tid = threadIdx.x;
  int block = blockIdx.x;
  int first = grainsize * tid;
  
  int2 range = mgpu::ComputeTaskRange(block, task, elements_per_group, count);
  
  // total is the sum of encountered elements. It's undefined on the first 
  // loop iteration.
  value_type total;
  bool totalDefined = false;
  
  // Loop through all tiles returned by ComputeTaskRange.
  while(range.x < range.y)
  {
    int count2 = min(elements_per_group, count - range.x);
    
    // Read tile data into register.
    input_type inputs[grainsize];
    mgpu::DeviceGlobalToReg<groupsize, grainsize>(count2, data_global + range.x, tid, inputs);
    
    if(Op::Commutative)
    {
      // This path exploits the commutative property of the operator.
      #pragma unroll
      for(int i = 0; i < grainsize; ++i)
      {
        int index = groupsize * i + tid;
        if(index < count2)
        {
          value_type x = inputs[i];
          total = (i || totalDefined) ? op.Plus(total, x) : x;
        }
      }
    }
    else
    {
      // Store the inputs to shared memory and read them back out in
      // thread order.
      mgpu::DeviceRegToShared<groupsize, grainsize>(elements_per_group, inputs, tid, shared.inputs);
      
      value_type x = op.Extract(op.Identity(), -1);			
      #pragma unroll
      for(int i = 0; i < grainsize; ++i)
      {
      	int index = first + i;
      	if(index < count2)
        {
      	  value_type y = op.Extract(shared.inputs[index], range.x + index);
      	  x = i ? op.Plus(x, y) : y;
      	}
      }
      __syncthreads();
      
      // Run a CTA-wide reduction
      x = R::Reduce(tid, x, shared.reduce, op);
      total = totalDefined ? op.Plus(total, x) : x;
    }
    
    range.x += elements_per_group;
    totalDefined = true;
  }  
  
  if(Op::Commutative)
  {
    // Run a CTA-wide reduction to sum the partials for each thread.
    total = R::Reduce(tid, total, shared.reduce, op);
  }
  
  if(!tid)
  {
    reduction_global[block] = total;
  }
}


template<typename InputIt, typename OutputIt, typename Op>
void IncScan(InputIt data_global, int count, OutputIt dest_global, Op op, mgpu::CudaContext& context)
{
  typedef typename Op::value_type value_type;
  typedef typename Op::result_type result_type;
  
  const int threshold_of_parallelism = 20000;

  if(count < threshold_of_parallelism)
  {
    const int size = 512;
    const int grainsize = 3;

    bulk::static_thread_group<size,grainsize> group;
    bulk::async(bulk::par(group, 1), inclusive_scan_n<size,grainsize>(), bulk::there, data_global, count, dest_global, thrust::plus<int>());
  }
  else
  {
    // Run the parallel raking reduce as an upsweep.
    const int groupsize1 = 128;
    const int grainsize1 = 7;
    typedef mgpu::LaunchBoxVT<groupsize1, grainsize1> Tuning;
    int2 launch = Tuning::GetLaunchParams(context);
    const int NV = launch.x * launch.y;
    
    int numTiles = MGPU_DIV_UP(count, NV);
    int numBlocks = std::min(context.NumSMs() * 25, numTiles);
    int2 task = mgpu::DivideTaskRange(numTiles, numBlocks);
    
    MGPU_MEM(value_type) reductionDevice = context.Malloc<value_type>(numBlocks);
    	
    // N loads
    my_KernelReduce<Tuning><<<numBlocks, launch.x>>>(data_global, count, task, reductionDevice->get(), op);
    
    // scan the sums to get the carries
    const unsigned int groupsize2 = 256;
    const unsigned int grainsize2 = 3;

    // XXX we could scatter the carries to the output instead of scanning in place
    //     this might simplify the next kernel
    bulk::static_thread_group<groupsize2,grainsize2> group2;
    bulk::async(bulk::par(group2,1), inclusive_scan_n<groupsize2,grainsize2>(), bulk::there, reductionDevice->get(), numBlocks, reductionDevice->get(), thrust::plus<int>());
    
    // do the downsweep - N loads, N stores
    bulk::static_thread_group<groupsize1,grainsize1> group1;
    bulk::async(bulk::par(group1,numBlocks), inclusive_downsweep<groupsize1,grainsize1>(), bulk::there, data_global, count, task, reductionDevice->get(), dest_global, thrust::plus<int>());
  }
}


template<typename InputIterator, typename OutputIterator>
OutputIterator my_inclusive_scan(InputIterator first, InputIterator last, OutputIterator result)
{
  mgpu::ContextPtr ctx = mgpu::CreateCudaDevice(0);

  ::IncScan(thrust::raw_pointer_cast(&*first),
            last - first,
            thrust::raw_pointer_cast(&*result),
            mgpu::ScanOp<mgpu::ScanOpTypeAdd,int>(),
            *ctx);

  return result + (last - first);
}


void my_scan(thrust::device_vector<T> *data)
{
  my_inclusive_scan(data->begin(), data->end(), data->begin());
}


void do_it(size_t n)
{
  thrust::host_vector<T> h_input(n);
  thrust::fill(h_input.begin(), h_input.end(), 1);

  thrust::host_vector<T> h_result(n);

  thrust::inclusive_scan(h_input.begin(), h_input.end(), h_result.begin());

  thrust::device_vector<T> d_input = h_input;
  thrust::device_vector<T> d_result(d_input.size());

  my_inclusive_scan(d_input.begin(), d_input.end(), d_result.begin());

  hipError_t error = hipDeviceSynchronize();

  if(error)
  {
    std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
  }

  assert(h_result == d_result);
}


template<typename InputIterator, typename OutputIterator>
OutputIterator mgpu_inclusive_scan(InputIterator first, InputIterator last, OutputIterator result)
{
  mgpu::ContextPtr ctx = mgpu::CreateCudaDevice(0);

  mgpu::Scan<mgpu::MgpuScanTypeInc>(thrust::raw_pointer_cast(&*first),
                                    last - first,
                                    thrust::raw_pointer_cast(&*result),
                                    mgpu::ScanOp<mgpu::ScanOpTypeAdd,int>(),
                                    (int*)0,
                                    false,
                                    *ctx);

  return result + (last - first);
}


void sean_scan(thrust::device_vector<T> *data)
{
  mgpu_inclusive_scan(data->begin(), data->end(), data->begin());
}


int main()
{
  for(size_t n = 1; n <= 1 << 20; n <<= 1)
  {
    std::cout << "Testing n = " << n << std::endl;
    do_it(n);
  }

  thrust::default_random_engine rng;
  for(int i = 0; i < 20; ++i)
  {
    size_t n = rng() % (1 << 20);
   
    std::cout << "Testing n = " << n << std::endl;
    do_it(n);
  }

  thrust::device_vector<T> vec(1 << 28);

  sean_scan(&vec);
  double sean_msecs = time_invocation_cuda(50, sean_scan, &vec);

  my_scan(&vec);
  double my_msecs = time_invocation_cuda(50, my_scan, &vec);

  std::cout << "Sean's time: " << sean_msecs << " ms" << std::endl;
  std::cout << "My time: " << my_msecs << " ms" << std::endl;

  std::cout << "My relative performance: " << sean_msecs / my_msecs << std::endl;

  return 0;
}

