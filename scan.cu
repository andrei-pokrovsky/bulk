#include "hip/hip_runtime.h"
#include <moderngpu.cuh>
#include <thrust/scan.h>
#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <cassert>
#include <iostream>
#include "time_invocation_cuda.hpp"
#include <thrust/detail/temporary_array.h>
#include <thrust/copy.h>
#include <bulk/bulk.hpp>


typedef int T;


template<unsigned int size, unsigned int grainsize>
struct inclusive_scan_n
{
  template<typename InputIterator, typename Size, typename OutputIterator, typename BinaryFunction>
  __device__ void operator()(bulk::static_thread_group<size,grainsize> &this_group, InputIterator first, Size n, OutputIterator result, BinaryFunction binary_op)
  {
    bulk::inclusive_scan(this_group, first, first + n, result, binary_op);
  }
};


template<unsigned int size, unsigned int grainsize>
struct exclusive_scan_n
{
  template<typename InputIterator, typename Size, typename OutputIterator, typename T, typename BinaryFunction>
  __device__ void operator()(bulk::static_thread_group<size,grainsize> &this_group, InputIterator first, Size n, OutputIterator result, T init, BinaryFunction binary_op)
  {
    bulk::exclusive_scan(this_group, first, first + n, result, init, binary_op);
  }
};


template<mgpu::MgpuScanType Type, typename InputIt, typename OutputIt, typename Op>
void IncScan(InputIt data_global, int count, OutputIt dest_global, Op op, mgpu::CudaContext& context)
{
  typedef typename Op::value_type value_type;
  typedef typename Op::result_type result_type;
  
  const int CutOff = 20000;

  if(count < CutOff)
  {
    const int size = 512;
    const int grainsize = 3;

    bulk::static_thread_group<size,grainsize> group;
    bulk::async(bulk::par(group, 1), inclusive_scan_n<size,grainsize>(), bulk::there, data_global, count, dest_global, thrust::plus<int>());
  }
  else
  {
    // Run the parallel raking reduce as an upsweep.
    const int NT = 128;
    const int VT = 7;
    typedef mgpu::LaunchBoxVT<NT, VT> Tuning;
    int2 launch = Tuning::GetLaunchParams(context);
    const int NV = launch.x * launch.y;
    
    int numTiles = MGPU_DIV_UP(count, NV);
    int numBlocks = std::min(context.NumSMs() * 25, numTiles);
    int2 task = mgpu::DivideTaskRange(numTiles, numBlocks);
    
    MGPU_MEM(value_type) reductionDevice = context.Malloc<value_type>(numBlocks + 1);
    	
    mgpu::KernelReduce<Tuning><<<numBlocks, launch.x>>>(data_global, count, task, reductionDevice->get(), op);
    
    // Run a parallel latency-oriented scan to reduce the spine of the 
    // raking reduction.
    const unsigned int groupsize2 = 256;
    const unsigned int grainsize2 = 3;

    bulk::static_thread_group<groupsize2,grainsize2> group;
    bulk::async(bulk::par(group,1), exclusive_scan_n<groupsize2,grainsize2>(), bulk::there, reductionDevice->get(), numBlocks, reductionDevice->get(), 0, thrust::plus<int>());
    
    // Run a raking scan as a downsweep.
    mgpu::KernelScanDownsweep<Tuning, Type><<<numBlocks, launch.x>>>(data_global, count, task, reductionDevice->get(), dest_global, false, op);
  }
}


template<typename InputIterator, typename OutputIterator>
OutputIterator my_inclusive_scan(InputIterator first, InputIterator last, OutputIterator result)
{
  mgpu::ContextPtr ctx = mgpu::CreateCudaDevice(0);

  ::IncScan<mgpu::MgpuScanTypeInc>(thrust::raw_pointer_cast(&*first),
                                   last - first,
                                   thrust::raw_pointer_cast(&*result),
                                   mgpu::ScanOp<mgpu::ScanOpTypeAdd,int>(),
                                   *ctx);

  return result + (last - first);
}


void my_scan(thrust::device_vector<T> *data)
{
  my_inclusive_scan(data->begin(), data->end(), data->begin());
}


void do_it(size_t n)
{
  thrust::host_vector<T> h_input(n);
  thrust::fill(h_input.begin(), h_input.end(), 1);

  thrust::host_vector<T> h_result(n);

  thrust::inclusive_scan(h_input.begin(), h_input.end(), h_result.begin());

  thrust::device_vector<T> d_input = h_input;
  thrust::device_vector<T> d_result(d_input.size());

  my_inclusive_scan(d_input.begin(), d_input.end(), d_result.begin());

  hipError_t error = hipDeviceSynchronize();

  if(error)
  {
    std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
  }

  assert(h_result == d_result);
}


template<typename InputIterator, typename OutputIterator>
OutputIterator mgpu_inclusive_scan(InputIterator first, InputIterator last, OutputIterator result)
{
  mgpu::ContextPtr ctx = mgpu::CreateCudaDevice(0);

  mgpu::Scan<mgpu::MgpuScanTypeInc>(thrust::raw_pointer_cast(&*first),
                                    last - first,
                                    thrust::raw_pointer_cast(&*result),
                                    mgpu::ScanOp<mgpu::ScanOpTypeAdd,int>(),
                                    (int*)0,
                                    false,
                                    *ctx);

  return result + (last - first);
}


void sean_scan(thrust::device_vector<T> *data)
{
  mgpu_inclusive_scan(data->begin(), data->end(), data->begin());
}


int main()
{
  for(size_t n = 1; n <= 1 << 20; n <<= 1)
  {
    std::cout << "Testing n = " << n << std::endl;
    do_it(n);
  }

  thrust::default_random_engine rng;
  for(int i = 0; i < 20; ++i)
  {
    size_t n = rng() % (1 << 20);
   
    std::cout << "Testing n = " << n << std::endl;
    do_it(n);
  }

  thrust::device_vector<T> vec(1 << 24);

  sean_scan(&vec);
  double sean_msecs = time_invocation_cuda(50, sean_scan, &vec);

  my_scan(&vec);
  double my_msecs = time_invocation_cuda(50, my_scan, &vec);

  std::cout << "Sean's time: " << sean_msecs << " ms" << std::endl;
  std::cout << "My time: " << my_msecs << " ms" << std::endl;

  std::cout << "My relative performance: " << sean_msecs / my_msecs << std::endl;

  return 0;
}

