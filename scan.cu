#include "hip/hip_runtime.h"
#include <thrust/scan.h>
#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <cassert>
#include <iostream>
#include "time_invocation_cuda.hpp"
#include <thrust/detail/temporary_array.h>
#include <thrust/detail/type_traits/function_traits.h>
#include <bulk/bulk.hpp>
#include "decomposition.hpp"


struct inclusive_scan_n
{
  template<typename ConcurrentGroup, typename InputIterator, typename Size, typename OutputIterator, typename T, typename BinaryFunction>
  __device__ void operator()(ConcurrentGroup &this_group, InputIterator first, Size n, OutputIterator result, T init, BinaryFunction binary_op)
  {
    bulk::inclusive_scan(this_group, first, first + n, result, init, binary_op);
  }
};

struct exclusive_scan_n
{
  template<typename ConcurrentGroup, typename InputIterator, typename Size, typename OutputIterator, typename T, typename BinaryFunction>
  __device__ void operator()(ConcurrentGroup &this_group, InputIterator first, Size n, OutputIterator result, T init, BinaryFunction binary_op)
  {
    bulk::exclusive_scan(this_group, first, first + n, result, init, binary_op);
  }
};


struct inclusive_downsweep
{
  template<typename ConcurrentGroup, typename RandomAccessIterator1, typename Decomposition, typename RandomAccessIterator2, typename RandomAccessIterator3, typename BinaryFunction>
  __device__ void operator()(ConcurrentGroup &this_group,
                             RandomAccessIterator1 first,
                             Decomposition decomp,
                             RandomAccessIterator2 carries_first,
                             RandomAccessIterator3 result,
                             BinaryFunction binary_op)
  {
    typename Decomposition::range range = decomp[this_group.index()];
  
    RandomAccessIterator1 last = first + range.second;
    first += range.first;
    result += range.first;
  
    typename thrust::iterator_value<RandomAccessIterator2>::type carry = carries_first[this_group.index()];

    bulk::inclusive_scan(this_group, first, last, result, carry, binary_op);
  }
};


struct accumulate_tiles
{
  template<typename ConcurrentGroup, typename RandomAccessIterator1, typename Decomposition, typename RandomAccessIterator2, typename BinaryFunction>
  __device__ void operator()(ConcurrentGroup &this_group,
                             RandomAccessIterator1 first,
                             Decomposition decomp,
                             RandomAccessIterator2 result,
                             BinaryFunction binary_op)
  {
    typedef typename thrust::iterator_value<RandomAccessIterator1>::type value_type;
    
    typename Decomposition::range range = decomp[this_group.index()];

    const bool commutative = thrust::detail::is_commutative<BinaryFunction>::value;

    // for a commutative accumulate, it's much faster to pass the last value as the init for some reason
    value_type init = commutative ? first[range.second-1] : *first;

    value_type sum = commutative ?
      bulk::accumulate(this_group, first + range.first, first + range.second - 1, init, binary_op) :
      bulk::accumulate(this_group, first + range.first + 1, first + range.second, init, binary_op);

    if(this_group.this_exec.index() == 0)
    {
      result[this_group.index()] = sum;
    } // end if
  } // end operator()
}; // end accumulate_tiles


template<typename RandomAccessIterator1, typename RandomAccessIterator2, typename T, typename BinaryFunction>
RandomAccessIterator2 inclusive_scan(RandomAccessIterator1 first, RandomAccessIterator1 last, RandomAccessIterator2 result, T init, BinaryFunction binary_op)
{
  typedef typename bulk::detail::scan_detail::scan_intermediate<
    RandomAccessIterator1,
    RandomAccessIterator2,
    BinaryFunction
  >::type intermediate_type;

  typedef typename thrust::iterator_difference<RandomAccessIterator1>::type Size;

  Size n = last - first;
  
  const Size threshold_of_parallelism = 20000;

  if(n < threshold_of_parallelism)
  {
    typedef bulk::detail::scan_detail::scan_buffer<512,3,RandomAccessIterator1,RandomAccessIterator2,BinaryFunction> heap_type;
    Size heap_size = sizeof(heap_type);
    bulk::async(bulk::par(bulk::con<512,3>(heap_size), 1), inclusive_scan_n(), bulk::root.this_exec, first, n, result, init, binary_op);
  } // end if
  else
  {
    // determined from empirical testing on k20c
    const int groupsize = sizeof(intermediate_type) <= sizeof(int) ? 128 : 256;
    const int grainsize = sizeof(intermediate_type) <= sizeof(int) ?   9 :   5;

    const Size tile_size = groupsize * grainsize;
    int num_tiles = (n + tile_size - 1) / tile_size;

    // 20 determined from empirical testing on k20c & GTX 480
    int subscription = 20;
    Size num_groups = thrust::min<Size>(subscription * bulk::concurrent_group<>::hardware_concurrency(), num_tiles);

    aligned_decomposition<Size> decomp(n, num_groups, tile_size);

    thrust::cuda::tag t;
    thrust::detail::temporary_array<intermediate_type,thrust::cuda::tag> carries(t, num_groups);
    	
    // Run the parallel raking reduce as an upsweep.
    // n loads + num_groups stores
    Size heap_size = groupsize * sizeof(intermediate_type);
    bulk::async(bulk::grid<groupsize,grainsize>(num_groups,heap_size), accumulate_tiles(), bulk::root.this_exec, first, decomp, carries.begin(), binary_op);
    
    // scan the sums to get the carries
    // num_groups loads + num_groups stores
    typedef bulk::detail::scan_detail::scan_buffer<256,3,RandomAccessIterator1,RandomAccessIterator2,BinaryFunction> heap_type2;
    heap_size = sizeof(heap_type2);
    bulk::async(bulk::con<256,3>(heap_size), exclusive_scan_n(), bulk::root.this_exec, carries.begin(), num_groups, carries.begin(), init, binary_op);

    // do the downsweep - n loads, n stores
    typedef bulk::detail::scan_detail::scan_buffer<
      groupsize,
      grainsize,
      RandomAccessIterator1,RandomAccessIterator2,BinaryFunction
    > heap_type3;
    heap_size = sizeof(heap_type3);
    bulk::async(bulk::grid<groupsize,grainsize>(num_groups,heap_size), inclusive_downsweep(), bulk::root.this_exec, first, decomp, carries.begin(), result, binary_op);
  } // end else

  return result + n;
} // end inclusive_scan()


template<typename T>
void my_scan(thrust::device_vector<T> *data, T init)
{
  ::inclusive_scan(data->begin(), data->end(), data->begin(), init, thrust::plus<T>());
}


template<typename T>
void validate(size_t n)
{
  thrust::host_vector<T> h_input(n);
  thrust::fill(h_input.begin(), h_input.end(), 1);

  thrust::host_vector<T> h_result(n);

  T init = 13;

  thrust::inclusive_scan(h_input.begin(), h_input.end(), h_result.begin());
  thrust::for_each(h_result.begin(), h_result.end(), thrust::placeholders::_1 += init);

  thrust::device_vector<T> d_input = h_input;
  thrust::device_vector<T> d_result(d_input.size());

  ::inclusive_scan(d_input.begin(), d_input.end(), d_result.begin(), init, thrust::plus<T>());

  hipError_t error = hipDeviceSynchronize();

  if(error)
  {
    std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
  }

  assert(h_result == d_result);
}


template<typename T>
void thrust_scan(thrust::device_vector<T> *data)
{
  thrust::inclusive_scan(data->begin(), data->end(), data->begin());
}


template<typename T>
void compare(size_t n = 1 << 28)
{
  thrust::device_vector<T> vec(n);

  thrust_scan(&vec);
  double thrust_msecs = time_invocation_cuda(50, thrust_scan<T>, &vec);

  my_scan(&vec, T(13));
  double my_msecs = time_invocation_cuda(50, my_scan<T>, &vec, 13);

  std::cout << "N: " << n << std::endl;
  std::cout << "  Thrust's time:                  " << thrust_msecs << " ms" << std::endl;
  std::cout << "  My time:                        " << my_msecs << " ms" << std::endl;
  std::cout << "  Performance relative to Thrust: " << thrust_msecs / my_msecs << std::endl;
  std::cout << std::endl;
}



int main()
{
  for(size_t n = 1; n <= 1 << 20; n <<= 1)
  {
    std::cout << "Testing n = " << n << std::endl;
    validate<int>(n);
  }

  thrust::default_random_engine rng;
  for(int i = 0; i < 20; ++i)
  {
    size_t n = rng() % (1 << 20);
   
    std::cout << "Testing n = " << n << std::endl;
    validate<int>(n);
  }

  std::cout << "32b int:" << std::endl;
  for(int i = 0; i < 28; ++i)
  {
    compare<int>(1 << i);
  }

  std::cout << "64b float:" << std::endl;
  for(int i = 0; i < 28; ++i)
  {
    compare<double>(1 << i);
  }

  return 0;
}

