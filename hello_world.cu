#include <iostream>
#include <cstdio>
#include "bulk_async.hpp"

struct hello
{
  __host__ __device__
  void operator()(unsigned int num_threads)
  {
    unsigned int thread_idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(thread_idx < num_threads)
    {
      printf("Hello world!\n");
    }
  }
};

int main()
{
  bulk_async::bulk_async(bulk_async::launch(1), hello(), 1);

  hipDeviceSynchronize();

  return 0;
}

