#include "hip/hip_runtime.h"
#include <iostream>
#include <moderngpu.cuh>
#include <thrust/device_vector.h>
#include <thrust/merge.h>
#include <thrust/sort.h>
#include <bulk/bulk.hpp>
#include "join_iterator.hpp"
#include "time_invocation_cuda.hpp"


template<std::size_t groupsize, std::size_t grainsize,
         typename RandomAccessIterator,
         typename Compare>
__device__
void bounded_inplace_merge(bulk::static_execution_group<groupsize,grainsize> &g,
                           RandomAccessIterator first, RandomAccessIterator middle, RandomAccessIterator last, Compare comp)
{
  int n1 = middle - first;
  int n2 = last - middle;

  // Run a merge path to find the start of the serial merge for each thread.
  int diag = grainsize * threadIdx.x;

  // XXX could invent an "inplace_merge_path" variant which didn't require redundant parameters n1 & middle
  int mp = mgpu::MergePath<mgpu::MgpuBoundsLower>(first, n1, middle, n2, diag, comp);
  
  // Compute the ranges of the sources in shared memory.
  int local_offset1 = mp;
  int local_offset2 = n1 + diag - mp;
  
  // Serial merge into register.
  typedef typename thrust::iterator_value<RandomAccessIterator>::type value_type;
  value_type local_result[grainsize];
  bulk::merge(bulk::bound<grainsize>(g.this_exec),
              first + local_offset1, middle,
              first + local_offset2, last,
              local_result,
              comp);

  g.wait();

  // local result back to source
  int local_offset = grainsize * threadIdx.x;

  // this is faster than getting the size from merge's result
  int local_size = thrust::max<int>(0, thrust::min<int>(grainsize, n1 + n2 - local_offset));
  bulk::copy_n(bulk::bound<grainsize>(g.this_exec), local_result, local_size, first + local_offset); 

  g.wait();
}


// XXX this is essentially a bounded version for group copy_n
//     the bound is NT * VT
template<std::size_t groupsize, std::size_t grainsize, typename RandomAccessIterator1, typename Size, typename RandomAccessIterator2>
__device__
RandomAccessIterator2 bounded_copy_n(bulk::static_execution_group<groupsize,grainsize> &g,
                                     RandomAccessIterator1 first,
                                     Size n,
                                     RandomAccessIterator2 result)
{
  typedef int size_type;

  size_type tid = g.this_exec.index();

  typedef typename thrust::iterator_value<RandomAccessIterator1>::type value_type;

  // XXX make this an uninitialized array
  value_type stage[grainsize];

  // avoid conditional accesses when possible
  if(groupsize * grainsize <= n)
  {
    #pragma unroll
    for(size_type i = 0; i < grainsize; ++i)
    {
      size_type src_idx = g.size() * i + tid;
      stage[i] = first[src_idx];
    } // end for i

    #pragma unroll
    for(size_type i = 0; i < grainsize; ++i)
    {
      size_type dst_idx = g.size() * i + tid;
      result[dst_idx] = stage[i];
    } // end for i
  } // end if
  else
  {
    #pragma unroll
    for(size_type i = 0; i < grainsize; ++i)
    {
      size_type src_idx = g.size() * i + tid;
      if(src_idx < n)
      {
        stage[i] = first[src_idx];
      } // end if
    } // end for

    #pragma unroll
    for(size_type i = 0; i < grainsize; ++i)
    {
      size_type dst_idx = g.size() * i + tid;
      if(dst_idx < n)
      {
        result[dst_idx] = stage[i];
      } // end if
    } // end for
  } // end else

  g.wait();

  return result + thrust::min<Size>(g.size() * grainsize, n);
}


template<std::size_t groupsize, std::size_t grainsize, typename KeysIt1, typename KeysIt2, typename KeyType, typename KeysIt3, typename Compare>
__device__
void my_DeviceMerge(KeysIt1 aKeys_global,
                    KeysIt2 bKeys_global,
                    int tid, int block,
                    int4 range,
                    KeyType* keys_shared,
                    KeysIt3 keys_global,
                    Compare comp)
{
  bulk::static_execution_group<groupsize,grainsize> exec;

  // Load the data into shared memory.
  int aCount = range.y - range.x;
  int bCount = range.w - range.z;

  bounded_copy_n(exec,
                 make_join_iterator(aKeys_global + range.x, aCount, bKeys_global + range.z),
                 aCount + bCount,
                 keys_shared);

  bounded_inplace_merge(exec, keys_shared, keys_shared + aCount, keys_shared + aCount + bCount, comp);
  
  // Store merged keys to global memory.
  // XXX this might be slightly faster with a bounded_copy_n
  bulk::copy_n(exec, keys_shared, aCount + bCount, keys_global + exec.size() * exec.grainsize() * block);
}


template<typename Tuning, bool HasValues, bool MergeSort, typename KeysIt1, 
	typename KeysIt2, typename KeysIt3, typename ValsIt1, typename ValsIt2,
	typename ValsIt3, typename Comp>
__global__
void my_KernelMerge(KeysIt1 aKeys_global, ValsIt1 aVals_global, int aCount,
                    KeysIt2 bKeys_global, ValsIt2 bVals_global, int bCount,
                    const int* mp_global,
                    int coop,
                    KeysIt3 keys_global, ValsIt3 vals_global,
                    Comp comp)
{
  typedef MGPU_LAUNCH_PARAMS Params;
  typedef typename std::iterator_traits<KeysIt1>::value_type KeyType;
  typedef typename std::iterator_traits<ValsIt1>::value_type ValType;
  
  const int NT = Params::NT;
  const int VT = Params::VT;

  __shared__ KeyType s_keys[NT * VT];
  
  int tid = threadIdx.x;
  int block = blockIdx.x;
  
  int4 range = mgpu::ComputeMergeRange(aCount, bCount, block, coop, NT * VT, mp_global);
  
  my_DeviceMerge<NT, VT>(aKeys_global,
                         bKeys_global,
                         tid,
                         block,
                         range,
                         s_keys, 
                         keys_global,
                         comp);
}


template<typename RandomAccessIterator1,
         typename RandomAccessIterator2,
         typename RandomAccessIterator3,
         typename Compare>
RandomAccessIterator3 my_merge(RandomAccessIterator1 first1,
                               RandomAccessIterator1 last1,
                               RandomAccessIterator2 first2,
                               RandomAccessIterator2 last2,
                               RandomAccessIterator3 result,
                               Compare comp)
{
  typedef typename thrust::iterator_value<RandomAccessIterator1>::type value_type;

  mgpu::ContextPtr ctx = mgpu::CreateCudaDevice(0);

  // XXX these seem to work well for K20c but could use some comprehensive tuning
  const int NT = 128 + 64;
  const int VT = 9;

  typedef mgpu::LaunchBoxVT<NT, VT> Tuning;
  int2 launch = Tuning::GetLaunchParams(*ctx);
  
  const int NV = launch.x * launch.y;

  // find partitions
  MGPU_MEM(int) partitionsDevice =
    mgpu::MergePathPartitions<mgpu::MgpuBoundsLower>(
      first1, last1 - first1,
      first2, last2 - first2,
      NV,
      0,
      comp,
      *ctx);

  // merge partitions
  int n = (last1 - first1) + (last2 - first2);

  // XXX it's easy to launch too many blocks this way
  //     we need to cap it and virtualize
  int num_blocks = (n + NV - 1) / NV;

  my_KernelMerge<Tuning, false, false><<<num_blocks, launch.x, 0, 0>>>
    (first1, (const int*)0, last1 - first1,
     first2, (const int*)0, last2 - first2, 
      partitionsDevice->get(), 0,
      result,
      (int*)0,
      comp);

  return result + n;
} // end merge()


template<typename T>
void my_merge(const thrust::device_vector<T> *a,
              const thrust::device_vector<T> *b,
              thrust::device_vector<T> *c)
{
  my_merge(a->begin(), a->end(),
           b->begin(), b->end(),
           c->begin(),
           thrust::less<T>());
}


template<typename T>
void sean_merge(const thrust::device_vector<T> *a,
                const thrust::device_vector<T> *b,
                thrust::device_vector<T> *c)
{
  mgpu::ContextPtr ctx = mgpu::CreateCudaDevice(0);
  mgpu::MergeKeys(a->begin(), a->size(),
                  b->begin(), b->size(),
                  c->begin(),
                  thrust::less<T>(),
                  *ctx);
}


template<typename T>
void thrust_merge(const thrust::device_vector<T> *a,
                  const thrust::device_vector<T> *b,
                  thrust::device_vector<T> *c)
{
  thrust::merge(a->begin(), a->end(),
                b->begin(), b->end(),
                c->begin(),
                thrust::less<T>());
}


template<typename T>
struct hash
{
  template<typename Integer>
  __device__ __device__
  T operator()(Integer x)
  {
    x = (x+0x7ed55d16) + (x<<12);
    x = (x^0xc761c23c) ^ (x>>19);
    x = (x+0x165667b1) + (x<<5);
    x = (x+0xd3a2646c) ^ (x<<9);
    x = (x+0xfd7046c5) + (x<<3);
    x = (x^0xb55a4f09) ^ (x>>16);
    return x;
  }
};


template<typename Vector>
void random_fill(Vector &vec)
{
  thrust::tabulate(vec.begin(), vec.end(), hash<typename Vector::value_type>());
}


template<typename T>
void compare(size_t n)
{
  thrust::device_vector<T> a(n / 2), b(n / 2);
  thrust::device_vector<T> c(n);

  random_fill(a);
  random_fill(b);

  thrust::sort(a.begin(), a.end());
  thrust::sort(b.begin(), b.end());

  my_merge(&a, &b, &c);
  double my_msecs = time_invocation_cuda(50, my_merge<T>, &a, &b, &c);

  sean_merge(&a, &b, &c);
  double sean_msecs = time_invocation_cuda(50, sean_merge<T>, &a, &b, &c);

  thrust_merge(&a, &b, &c);
  double thrust_msecs = time_invocation_cuda(50, thrust_merge<T>, &a, &b, &c);

  std::cout << "Sean's time: " << sean_msecs << " ms" << std::endl;
  std::cout << "Thrust's time: " << thrust_msecs << " ms" << std::endl;
  std::cout << "My time:       " << my_msecs << " ms" << std::endl;

  std::cout << "Performance relative to Sean: " << sean_msecs / my_msecs << std::endl;
  std::cout << "Performance relative to Thrust: " << thrust_msecs / my_msecs << std::endl;
}


template<typename T>
void validate(size_t n)
{
  thrust::device_vector<T> a(n / 2), b(n / 2);
  thrust::device_vector<T> c(n);

  random_fill(a);
  random_fill(b);

  thrust::sort(a.begin(), a.end());
  thrust::sort(b.begin(), b.end());

  thrust::device_vector<T> ref(n);
  thrust::merge(a.begin(), a.end(), b.begin(), b.end(), ref.begin());

  my_merge(&a, &b, &c);

  std::cout << "CUDA error: " << hipGetErrorString(hipDeviceSynchronize()) << std::endl;

  assert(c == ref);
}


int main()
{
  size_t n = 123456789;

  validate<int>(n);

  std::cout << "Large input: " << std::endl;
  std::cout << "int: " << std::endl;
  compare<int>(n);

  std::cout << "float: " << std::endl;
  compare<float>(n);

  std::cout << "double: " << std::endl;
  compare<double>(n);
  std::cout << std::endl;

  return 0;
}

