#include "hip/hip_runtime.h"
#include <iostream>
#include <moderngpu.cuh>
#include <thrust/device_vector.h>
#include <thrust/merge.h>
#include <thrust/sort.h>
#include "time_invocation_cuda.hpp"


template<int NT, int VT, typename It1, typename It2, typename T, typename Comp>
__device__
void my_DeviceMergeKeysIndices(It1 a_global, It2 b_global, int4 range, int tid, T* keys_shared, T* results, int* indices, Comp comp)
{
  int a0 = range.x;
  int a1 = range.y;
  int b0 = range.z;
  int b1 = range.w;
  int aCount = a1 - a0;
  int bCount = b1 - b0;
  
  // Load the data into shared memory.
  mgpu::DeviceLoad2ToShared<NT, VT, VT>(a_global + a0, aCount, b_global + b0, bCount, tid, keys_shared);
  
  // Run a merge path to find the start of the serial merge for each thread.
  int diag = VT * tid;
  int mp = mgpu::MergePath<mgpu::MgpuBoundsLower>(keys_shared, aCount, keys_shared + aCount, bCount, diag, comp);
  
  // Compute the ranges of the sources in shared memory.
  int a0tid = mp;
  int a1tid = aCount;
  int b0tid = aCount + diag - mp;
  int b1tid = aCount + bCount;
  
  // Serial merge into register.
  mgpu::SerialMerge<VT, true>(keys_shared, a0tid, a1tid, b0tid, b1tid, results, indices, comp);
}


template<int NT, int VT, typename KeysIt1, typename KeysIt2, typename KeysIt3, typename KeyType, typename Comp>
__device__
void my_DeviceMerge(KeysIt1 aKeys_global,
                    KeysIt2 bKeys_global,
                    int tid, int block,
                    int4 range,
                    KeyType* keys_shared,
                    KeysIt3 keys_global,
                    Comp comp)
{
  KeyType results[VT];
  int indices[VT];
  my_DeviceMergeKeysIndices<NT, VT>(aKeys_global, bKeys_global, range, tid, keys_shared, results, indices, comp);
  
  // Store merge results back to shared memory.
  mgpu::DeviceThreadToShared<VT>(results, tid, keys_shared);
  
  // Store merged keys to global memory.
  int aCount = range.y - range.x;
  int bCount = range.w - range.z;
  mgpu::DeviceSharedToGlobal<NT, VT>(aCount + bCount, keys_shared, tid, keys_global + NT * VT * block);
}


template<typename Tuning, bool HasValues, bool MergeSort, typename KeysIt1, 
	typename KeysIt2, typename KeysIt3, typename ValsIt1, typename ValsIt2,
	typename ValsIt3, typename Comp>
__global__
void KernelMerge(KeysIt1 aKeys_global, ValsIt1 aVals_global, int aCount,
                 KeysIt2 bKeys_global, ValsIt2 bVals_global, int bCount,
                 const int* mp_global,
                 int coop,
                 KeysIt3 keys_global, ValsIt3 vals_global,
                 Comp comp)
{
  typedef MGPU_LAUNCH_PARAMS Params;
  typedef typename std::iterator_traits<KeysIt1>::value_type KeyType;
  typedef typename std::iterator_traits<ValsIt1>::value_type ValType;
  
  const int NT = Params::NT;
  const int VT = Params::VT;
  const int NV = NT * VT;
  union Shared {
  	KeyType keys[NT * (VT + 1)];
  	int indices[NV];
  };
  __shared__ Shared shared;
  
  int tid = threadIdx.x;
  int block = blockIdx.x;
  
  int4 range = mgpu::ComputeMergeRange(aCount, bCount, block, coop, NT * VT, mp_global);
  
  my_DeviceMerge<NT, VT>(aKeys_global,
                         bKeys_global,
                         tid,
                         block,
                         range,
                         shared.keys, 
                         keys_global,
                         comp);
}


template<typename RandomAccessIterator1,
         typename RandomAccessIterator2,
         typename RandomAccessIterator3,
         typename Compare>
RandomAccessIterator3 my_merge(RandomAccessIterator1 first1,
                               RandomAccessIterator1 last1,
                               RandomAccessIterator2 first2,
                               RandomAccessIterator2 last2,
                               RandomAccessIterator3 result,
                               Compare comp)
{
  typedef typename thrust::iterator_value<RandomAccessIterator1>::type value_type;

  mgpu::ContextPtr ctx = mgpu::CreateCudaDevice(0);

  const int NT = 128;
  const int VT = 11;
  typedef mgpu::LaunchBoxVT<NT, VT> Tuning;
  int2 launch = Tuning::GetLaunchParams(*ctx);
  
  const int NV = launch.x * launch.y;

  // find partitions
  MGPU_MEM(int) partitionsDevice =
    mgpu::MergePathPartitions<mgpu::MgpuBoundsLower>(
      first1, last1 - first1,
      first2, last2 - first2,
      NV,
      0,
      comp,
      *ctx);

  // merge partitions
  int n = (last1 - first1) + (last2 - first2);
  int num_blocks = (n + NV - 1) / NV;
  mgpu::KernelMerge<Tuning, false, false><<<num_blocks, launch.x, 0, 0>>>
    (first1, (const int*)0, last1 - first1,
     first2, (const int*)0, last2 - first2, 
      partitionsDevice->get(), 0,
      result,
      (int*)0,
      comp);

  return result + n;
} // end merge()


template<typename T>
void my_merge(const thrust::device_vector<T> *a,
              const thrust::device_vector<T> *b,
              thrust::device_vector<T> *c)
{
  my_merge(a->begin(), a->end(),
           b->begin(), b->end(),
           c->begin(),
           thrust::less<T>());
}


template<typename T>
void sean_merge(const thrust::device_vector<T> *a,
                const thrust::device_vector<T> *b,
                thrust::device_vector<T> *c)
{
  mgpu::ContextPtr ctx = mgpu::CreateCudaDevice(0);
  mgpu::MergeKeys(a->begin(), a->size(),
                  b->begin(), b->size(),
                  c->begin(),
                  thrust::less<T>(),
                  *ctx);
}


template<typename T>
void thrust_merge(const thrust::device_vector<T> *a,
                  const thrust::device_vector<T> *b,
                  thrust::device_vector<T> *c)
{
  thrust::merge(a->begin(), a->end(),
                b->begin(), b->end(),
                c->begin(),
                thrust::less<T>());
}


template<typename T>
struct hash
{
  template<typename Integer>
  __device__ __device__
  T operator()(Integer x)
  {
    x = (x+0x7ed55d16) + (x<<12);
    x = (x^0xc761c23c) ^ (x>>19);
    x = (x+0x165667b1) + (x<<5);
    x = (x+0xd3a2646c) ^ (x<<9);
    x = (x+0xfd7046c5) + (x<<3);
    x = (x^0xb55a4f09) ^ (x>>16);
    return x;
  }
};


template<typename Vector>
void random_fill(Vector &vec)
{
  thrust::tabulate(vec.begin(), vec.end(), hash<typename Vector::value_type>());
}


template<typename T>
void compare(size_t n)
{
  thrust::device_vector<T> a(n / 2), b(n / 2);
  thrust::device_vector<T> c(n);

  random_fill(a);
  random_fill(b);

  thrust::sort(a.begin(), a.end());
  thrust::sort(b.begin(), b.end());

  my_merge(&a, &b, &c);
  double my_msecs = time_invocation_cuda(50, my_merge<T>, &a, &b, &c);

  sean_merge(&a, &b, &c);
  double sean_msecs = time_invocation_cuda(50, sean_merge<T>, &a, &b, &c);

  thrust_merge(&a, &b, &c);
  double thrust_msecs = time_invocation_cuda(50, thrust_merge<T>, &a, &b, &c);

  std::cout << "Sean's time: " << sean_msecs << " ms" << std::endl;
  std::cout << "Thrust's time: " << thrust_msecs << " ms" << std::endl;
  std::cout << "My time:       " << my_msecs << " ms" << std::endl;

  std::cout << "Performance relative to Sean: " << sean_msecs / my_msecs << std::endl;
  std::cout << "Performance relative to Thrust: " << thrust_msecs / my_msecs << std::endl;
}


int main()
{
  size_t n = 123456789;

  thrust::device_vector<int> a(n / 2), b(n / 2);
  thrust::device_vector<int> c(n);

  random_fill(a);
  random_fill(b);
  thrust::sort(a.begin(), a.end());
  thrust::sort(b.begin(), b.end());

  my_merge(&a, &b, &c);

  thrust::device_vector<int> ref(n);

  thrust_merge(&a, &b, &ref);

  assert(c == ref);

  std::cout << "Large input: " << std::endl;
  std::cout << "int: " << std::endl;
  compare<int>(n);

  std::cout << "float: " << std::endl;
  compare<float>(n);

  std::cout << "double: " << std::endl;
  compare<double>(n);
  std::cout << std::endl;

  return 0;
}


