#include "hip/hip_runtime.h"
#include <iostream>
#include <moderngpu.cuh>
#include <thrust/device_vector.h>
#include <thrust/merge.h>
#include <thrust/sort.h>
#include <bulk/bulk.hpp>
#include "time_invocation_cuda.hpp"


template<int VT, typename RandomAccessIterator1, typename RandomAccessIterator2, typename RandomAccessIterator3, typename Compare>
__device__
void my_SerialMerge(RandomAccessIterator1 first1, RandomAccessIterator1 last1,
                    RandomAccessIterator2 first2, RandomAccessIterator2 last2,
                    RandomAccessIterator3 result,
                    Compare comp)
{ 
  typedef typename thrust::iterator_value<RandomAccessIterator1>::type value_type1;
  typedef typename thrust::iterator_value<RandomAccessIterator2>::type value_type2;

  int n1 = last1 - first1;
  int idx1 = 0;

  int n2 = last2 - first2;
  int idx2 = 0;

  value_type1 a = *first1;
  value_type2 b = *first2;

  bool exhausted2 = idx2 >= n2;
  
  #pragma unroll
  for(int i = 0; i < VT; ++i)
  {
    //bool p = (idx2 >= n2) || ((idx1 < n1) && !comp(b, a));
    bool p = exhausted2 || ((idx1 < n1) && !comp(b, a));
    
    if(idx1 < n1 || !exhausted2)
    {
      result[i] = p ? a : b;
    }
    
    if(p)
    {
      ++idx1;

      // XXX making this a conditional load is pretty expensive for some reason
      a = first1[idx1];
    }
    else
    {
      ++idx2;

      if(idx2 >= n2) exhausted2 = true;

      // XXX making this a conditional load is pretty expensive for some reason
      b = first2[idx2];
    }
  }
}


template<std::size_t bound,
         typename InputIterator1,
         typename InputIterator2,
         typename OutputIterator,
         typename Compare>
__device__
OutputIterator bounded_merge(InputIterator1 first1, InputIterator1 last1,
                             InputIterator2 first2, InputIterator2 last2,
                             OutputIterator result,
                             Compare comp)
{
  typedef typename thrust::iterator_value<InputIterator1>::type value_type1;
  typedef typename thrust::iterator_value<InputIterator2>::type value_type2;

  int n1 = last1 - first1;
  int idx1 = 0;

  int n2 = last2 - first2;
  int idx2 = 0;

  // XXX these should be uninitialized
  value_type1 a;
  value_type2 b;

  if(n1)
  {
    a = first1[0];
  }

  if(n2)
  {
    b = first2[0];
  }

  #pragma unroll
  for(int i = 0; i < bound; ++i)
  {
    // 4 cases:
    // 0. both ranges are exhausted
    // 1. range 1 is exhausted
    // 2. range 2 is exhausted
    // 3. neither range is exhausted

    const bool exhausted1 = idx1 >= n1;
    const bool exhausted2 = idx2 >= n2;

    if(exhausted1 && exhausted2)
    {
      ;
    } // end if
    else if(exhausted1)
    {
      result[i] = b;
      ++idx2;
    } // end else if
    else if(exhausted2)
    {
      result[i] = a;
      ++idx1;
    } // end else if
    else
    {
      if(!comp(b,a))
      {
        result[i] = a;
        ++idx1;

        if(idx1 < n1)
        {
          a = first1[idx1];
        } // end if
      } // end if
      else
      {
        result[i] = b;
        ++idx2;

        if(idx2 < n2)
        {
          b = first2[idx2];
        } // end if
      } // end else
    } // end else
  } // end for i

  return result;
} // end bounded_merge


template<std::size_t groupsize, std::size_t grainsize,
         typename RandomAccessIterator,
         typename Compare>
__device__
void bounded_inplace_merge(RandomAccessIterator first, RandomAccessIterator middle, RandomAccessIterator last, Compare comp)
{
  bulk::static_execution_group<groupsize,grainsize> g;

  int n1 = middle - first;
  int n2 = last - middle;

  // Run a merge path to find the start of the serial merge for each thread.
  int diag = grainsize * threadIdx.x;

  // XXX could invent an "inplace_merge_path" variant which didn't require redundant parameters n1 & middle
  int mp = mgpu::MergePath<mgpu::MgpuBoundsLower>(first, n1, middle, n2, diag, comp);
  
  // Compute the ranges of the sources in shared memory.
  int local_offset1 = mp;
  int local_offset2 = n1 + diag - mp;
  
  // Serial merge into register.
  typedef typename thrust::iterator_value<RandomAccessIterator>::type value_type;
  value_type local_result[grainsize];
  my_SerialMerge<grainsize>(first + local_offset1, middle,
                            first + local_offset2, last,
                            local_result,
                            comp);

  g.wait();

  // local result back to source
  int local_offset = grainsize * threadIdx.x;
  int local_size = thrust::min<int>(grainsize, n1 + n2 - local_offset);
  bulk::copy_n(bulk::bound<grainsize>(g), local_result, local_size, first + local_offset); 

  g.wait();
}


template<int NT, int VT, typename KeysIt1, typename KeysIt2, typename KeysIt3, typename KeyType, typename Comp>
__device__
void my_DeviceMerge(KeysIt1 aKeys_global,
                    KeysIt2 bKeys_global,
                    int tid, int block,
                    int4 range,
                    KeyType* keys_shared,
                    KeysIt3 keys_global,
                    Comp comp)
{
  // Load the data into shared memory.
  int aCount = range.y - range.x;
  int bCount = range.w - range.z;
  mgpu::DeviceLoad2ToShared<NT, VT, VT>(aKeys_global + range.x, aCount, bKeys_global + range.z, bCount, tid, keys_shared);

  bounded_inplace_merge<NT,VT>(keys_shared, keys_shared + aCount, keys_shared + aCount + bCount, comp);
  
  // Store merged keys to global memory.
  mgpu::DeviceSharedToGlobal<NT, VT>(aCount + bCount, keys_shared, tid, keys_global + NT * VT * block);
}


template<typename Tuning, bool HasValues, bool MergeSort, typename KeysIt1, 
	typename KeysIt2, typename KeysIt3, typename ValsIt1, typename ValsIt2,
	typename ValsIt3, typename Comp>
__global__
void my_KernelMerge(KeysIt1 aKeys_global, ValsIt1 aVals_global, int aCount,
                    KeysIt2 bKeys_global, ValsIt2 bVals_global, int bCount,
                    const int* mp_global,
                    int coop,
                    KeysIt3 keys_global, ValsIt3 vals_global,
                    Comp comp)
{
  typedef MGPU_LAUNCH_PARAMS Params;
  typedef typename std::iterator_traits<KeysIt1>::value_type KeyType;
  typedef typename std::iterator_traits<ValsIt1>::value_type ValType;
  
  const int NT = Params::NT;
  const int VT = Params::VT;
  union Shared {
  	KeyType keys[NT * (VT + 1)];
  };
  __shared__ Shared shared;
  
  int tid = threadIdx.x;
  int block = blockIdx.x;
  
  int4 range = mgpu::ComputeMergeRange(aCount, bCount, block, coop, NT * VT, mp_global);
  
  my_DeviceMerge<NT, VT>(aKeys_global,
                         bKeys_global,
                         tid,
                         block,
                         range,
                         shared.keys, 
                         keys_global,
                         comp);
}


template<typename RandomAccessIterator1,
         typename RandomAccessIterator2,
         typename RandomAccessIterator3,
         typename Compare>
RandomAccessIterator3 my_merge(RandomAccessIterator1 first1,
                               RandomAccessIterator1 last1,
                               RandomAccessIterator2 first2,
                               RandomAccessIterator2 last2,
                               RandomAccessIterator3 result,
                               Compare comp)
{
  typedef typename thrust::iterator_value<RandomAccessIterator1>::type value_type;

  mgpu::ContextPtr ctx = mgpu::CreateCudaDevice(0);

  const int NT = 128;
  const int VT = 11;
  typedef mgpu::LaunchBoxVT<NT, VT> Tuning;
  int2 launch = Tuning::GetLaunchParams(*ctx);
  
  const int NV = launch.x * launch.y;

  // find partitions
  MGPU_MEM(int) partitionsDevice =
    mgpu::MergePathPartitions<mgpu::MgpuBoundsLower>(
      first1, last1 - first1,
      first2, last2 - first2,
      NV,
      0,
      comp,
      *ctx);

  // merge partitions
  int n = (last1 - first1) + (last2 - first2);
  int num_blocks = (n + NV - 1) / NV;
  my_KernelMerge<Tuning, false, false><<<num_blocks, launch.x, 0, 0>>>
    (first1, (const int*)0, last1 - first1,
     first2, (const int*)0, last2 - first2, 
      partitionsDevice->get(), 0,
      result,
      (int*)0,
      comp);

  return result + n;
} // end merge()


template<typename T>
void my_merge(const thrust::device_vector<T> *a,
              const thrust::device_vector<T> *b,
              thrust::device_vector<T> *c)
{
  my_merge(a->begin(), a->end(),
           b->begin(), b->end(),
           c->begin(),
           thrust::less<T>());
}


template<typename T>
void sean_merge(const thrust::device_vector<T> *a,
                const thrust::device_vector<T> *b,
                thrust::device_vector<T> *c)
{
  mgpu::ContextPtr ctx = mgpu::CreateCudaDevice(0);
  mgpu::MergeKeys(a->begin(), a->size(),
                  b->begin(), b->size(),
                  c->begin(),
                  thrust::less<T>(),
                  *ctx);
}


template<typename T>
void thrust_merge(const thrust::device_vector<T> *a,
                  const thrust::device_vector<T> *b,
                  thrust::device_vector<T> *c)
{
  thrust::merge(a->begin(), a->end(),
                b->begin(), b->end(),
                c->begin(),
                thrust::less<T>());
}


template<typename T>
struct hash
{
  template<typename Integer>
  __device__ __device__
  T operator()(Integer x)
  {
    x = (x+0x7ed55d16) + (x<<12);
    x = (x^0xc761c23c) ^ (x>>19);
    x = (x+0x165667b1) + (x<<5);
    x = (x+0xd3a2646c) ^ (x<<9);
    x = (x+0xfd7046c5) + (x<<3);
    x = (x^0xb55a4f09) ^ (x>>16);
    return x;
  }
};


template<typename Vector>
void random_fill(Vector &vec)
{
  thrust::tabulate(vec.begin(), vec.end(), hash<typename Vector::value_type>());
}


template<typename T>
void compare(size_t n)
{
  thrust::device_vector<T> a(n / 2), b(n / 2);
  thrust::device_vector<T> c(n);

  random_fill(a);
  random_fill(b);

  thrust::sort(a.begin(), a.end());
  thrust::sort(b.begin(), b.end());

  my_merge(&a, &b, &c);
  double my_msecs = time_invocation_cuda(50, my_merge<T>, &a, &b, &c);

  sean_merge(&a, &b, &c);
  double sean_msecs = time_invocation_cuda(50, sean_merge<T>, &a, &b, &c);

  thrust_merge(&a, &b, &c);
  double thrust_msecs = time_invocation_cuda(50, thrust_merge<T>, &a, &b, &c);

  std::cout << "Sean's time: " << sean_msecs << " ms" << std::endl;
  std::cout << "Thrust's time: " << thrust_msecs << " ms" << std::endl;
  std::cout << "My time:       " << my_msecs << " ms" << std::endl;

  std::cout << "Performance relative to Sean: " << sean_msecs / my_msecs << std::endl;
  std::cout << "Performance relative to Thrust: " << thrust_msecs / my_msecs << std::endl;
}


int main()
{
  size_t n = 123456789;

  thrust::device_vector<int> a(n / 2), b(n / 2);
  thrust::device_vector<int> c(n);

  random_fill(a);
  random_fill(b);
  thrust::sort(a.begin(), a.end());
  thrust::sort(b.begin(), b.end());

  my_merge(&a, &b, &c);

  thrust::device_vector<int> ref(n);

  thrust_merge(&a, &b, &ref);

  assert(c == ref);

  std::cout << "Large input: " << std::endl;
  std::cout << "int: " << std::endl;
  compare<int>(n);

  std::cout << "float: " << std::endl;
  compare<float>(n);

  std::cout << "double: " << std::endl;
  compare<double>(n);
  std::cout << std::endl;

  return 0;
}


