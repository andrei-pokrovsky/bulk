#include "hip/hip_runtime.h"
#include <iostream>
#include <moderngpu.cuh>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/detail/swap.h>
#include "time_invocation_cuda.hpp"


template<int i, int bound>
struct stable_odd_even_transpose_sort_impl
{
  template<typename RandomAccessIterator1, typename RandomAccessIterator2, typename Compare>
  static __device__
  void sort(RandomAccessIterator1 keys, RandomAccessIterator2 values, Compare comp)
  {
    #pragma unroll
    for(int j = 1 & i; j < bound - 1; j += 2)
    {
      if(comp(keys[j + 1], keys[j]))
      {
        using thrust::swap;

      	swap(keys[j], keys[j + 1]);
      	swap(values[j], values[j + 1]);
      }
    }

    stable_odd_even_transpose_sort_impl<i + 1, bound>::sort(keys, values, comp);
  }
};


template<int i> struct stable_odd_even_transpose_sort_impl<i, i>
{
  template<typename RandomAccessIterator1, typename RandomAccessIterator2, typename Compare>
  static __device__ void sort(RandomAccessIterator1 keys, RandomAccessIterator2 values, Compare comp) { }
};


template<int bound, typename RandomAccessIterator1, typename RandomAccessIterator2, typename Compare>
__device__
void OddEvenTransposeSort(RandomAccessIterator1 keys, RandomAccessIterator2 values, Compare comp)
{
  stable_odd_even_transpose_sort_impl<0, bound>::sort(keys, values, comp);
}


template<int NT, int VT, typename KeyType, typename ValType, typename Comp>
__device__
void CTAMergesort(KeyType threadKeys[VT], ValType threadValues[VT], KeyType* keys_shared, ValType* values_shared, int count, int tid, Comp comp)
{
  // Stable sort the keys in the thread.
  if(VT * tid < count)
  {
    ::OddEvenTransposeSort<VT>(threadKeys, threadValues, comp);
  }
  
  // Store the locally sorted keys into shared memory.
  mgpu::DeviceThreadToShared<VT>(threadKeys, tid, keys_shared);
  
  // Recursively merge lists until the entire CTA is sorted.
  mgpu::CTABlocksortLoop<NT, VT, true>(threadValues, keys_shared, values_shared, tid, count, comp);
}


template<typename Tuning, typename KeyIt1, typename KeyIt2, typename ValIt1, typename ValIt2, typename Comp>
__global__ void KernelBlocksort(KeyIt1 keysSource_global, ValIt1 valsSource_global, int count, KeyIt2 keysDest_global, ValIt2 valsDest_global, Comp comp)
{
  typedef MGPU_LAUNCH_PARAMS Params;
  typedef typename std::iterator_traits<KeyIt1>::value_type KeyType;
  typedef typename std::iterator_traits<ValIt1>::value_type ValType;
  
  const int groupsize = Params::NT;
  const int grainsize = Params::VT;
  const int tile_size = groupsize * grainsize;
  union Shared
  {
    KeyType keys[groupsize * (grainsize + 1)];
    ValType values[tile_size];
  };
  __shared__ Shared shared;
  
  int tid = threadIdx.x;
  int block = blockIdx.x;
  int gid = tile_size * block;
  int count2 = min(tile_size, count - gid);
  
  // Load the values into thread order.
  ValType threadValues[grainsize];
  mgpu::DeviceGlobalToShared<groupsize, grainsize>(count2, valsSource_global + gid, tid, shared.values);
  mgpu::DeviceSharedToThread<grainsize>(shared.values, tid, threadValues);
  
  // Load keys into shared memory and transpose into register in thread order.
  KeyType threadKeys[grainsize];
  mgpu::DeviceGlobalToShared<groupsize, grainsize>(count2, keysSource_global + gid, tid, shared.keys);
  mgpu::DeviceSharedToThread<grainsize>(shared.keys, tid, threadKeys);
  
  // If we're in the last tile, set the uninitialized keys for the thread with
  // a partial number of keys.
  int first = grainsize * tid;
  if(first + grainsize > count2 && first < count2)
  {
    KeyType maxKey = threadKeys[0];
    #pragma unroll
    for(int i = 1; i < grainsize; ++i)
    {
      if(first + i < count2)
      {
      	maxKey = comp(maxKey, threadKeys[i]) ? threadKeys[i] : maxKey;
      }
    }
    
    // Fill in the uninitialized elements with max key.
    #pragma unroll
    for(int i = 0; i < grainsize; ++i)
    {
      if(first + i >= count2) threadKeys[i] = maxKey;
    }
  }
  
  ::CTAMergesort<groupsize, grainsize, true>(threadKeys, threadValues, shared.keys, shared.values, count2, tid, comp);
  
  // Store the sorted keys to global.
  mgpu::DeviceSharedToGlobal<groupsize, grainsize>(count2, shared.keys, tid, keysDest_global + gid);
  mgpu::DeviceThreadToShared<grainsize>(threadValues, tid, shared.values);
  mgpu::DeviceSharedToGlobal<groupsize, grainsize>(count2, shared.values, tid, valsDest_global + gid);
}


template<typename KeyType, typename ValType, typename Comp>
void MergesortPairs(KeyType* keys_global, ValType* values_global, int count, Comp comp, mgpu::CudaContext& context)
{
  const int NT = 256;
  const int VT = 11;
  typedef mgpu::LaunchBoxVT<NT, VT> Tuning;
  int2 launch = Tuning::GetLaunchParams(context);
  
  const int NV = launch.x * launch.y;
  int numBlocks = MGPU_DIV_UP(count, NV);
  int numPasses = mgpu::FindLog2(numBlocks, true);
  
  MGPU_MEM(KeyType) keysDestDevice = context.Malloc<KeyType>(count);
  MGPU_MEM(ValType) valsDestDevice = context.Malloc<ValType>(count);
  KeyType* keysSource = keys_global;
  KeyType* keysDest = keysDestDevice->get();
  ValType* valsSource = values_global;
  ValType* valsDest = valsDestDevice->get();
  
  mgpu::KernelBlocksort<Tuning, true><<<numBlocks, launch.x, 0, context.Stream()>>>(keysSource, valsSource, count, (1 & numPasses) ? keysDest : keysSource, (1 & numPasses) ? valsDest : valsSource, comp);

  if(1 & numPasses)
  {
    std::swap(keysSource, keysDest);
    std::swap(valsSource, valsDest);
  }
  
  for(int pass = 0; pass < numPasses; ++pass) 
  {
    int coop = 2<< pass;
    MGPU_MEM(int) partitionsDevice = mgpu::MergePathPartitions<mgpu::MgpuBoundsLower>(keysSource, count, keysSource, 0, NV, coop, comp, context);
    
    mgpu::KernelMerge<Tuning, true, true><<<numBlocks, launch.x, 0, context.Stream()>>>(keysSource, valsSource, count, keysSource, valsSource, 0, partitionsDevice->get(), coop, keysDest, valsDest, comp);

    std::swap(keysDest, keysSource);
    std::swap(valsDest, valsSource);
  }
}


template<typename RandomAccessIterator1, typename RandomAccessIterator2, typename Compare>
void my_sort_by_key_(RandomAccessIterator1 keys_first, RandomAccessIterator1 keys_last, RandomAccessIterator2 values_first, Compare comp)
{
  mgpu::ContextPtr ctx = mgpu::CreateCudaDevice(0);
  ::MergesortPairs(thrust::raw_pointer_cast(&*keys_first),
                   thrust::raw_pointer_cast(&*values_first),
                   keys_last - keys_first,
                   comp,
                   *ctx);
}


template<typename T>
void my_sort_by_key(const thrust::device_vector<T> *unsorted_keys,
                    const thrust::device_vector<T> *unsorted_values,
                    thrust::device_vector<T> *sorted_keys,
                    thrust::device_vector<T> *sorted_values)
{
  *sorted_keys = *unsorted_keys;
  *sorted_values = *unsorted_values;
  my_sort_by_key_(sorted_keys->begin(), sorted_keys->end(), sorted_values->begin(), thrust::less<T>());
}


template<typename T>
void sean_sort_by_key(const thrust::device_vector<T> *unsorted_keys,
                      const thrust::device_vector<T> *unsorted_values,                    
                      thrust::device_vector<T> *sorted_keys,
                      thrust::device_vector<T> *sorted_values)
{
  *sorted_keys = *unsorted_keys;
  *sorted_values = *unsorted_values;
  mgpu::ContextPtr ctx = mgpu::CreateCudaDevice(0);
  mgpu::MergesortPairs(thrust::raw_pointer_cast(sorted_keys->data()),
                       thrust::raw_pointer_cast(sorted_values->data()),
                       sorted_keys->size(),
                       thrust::less<T>(),
                       *ctx);
}


template<typename T>
void thrust_sort_by_key(const thrust::device_vector<T> *unsorted_keys,
                        const thrust::device_vector<T> *unsorted_values,
                        thrust::device_vector<T> *sorted_keys,
                        thrust::device_vector<T> *sorted_values)
{
  *sorted_keys = *unsorted_keys;
  *sorted_values = *unsorted_values;
  thrust::sort_by_key(sorted_keys->begin(), sorted_keys->end(), sorted_values->begin(), thrust::less<T>());
}


template<typename T>
struct hash
{
  template<typename Integer>
  __device__ __device__
  T operator()(Integer x)
  {
    x = (x+0x7ed55d16) + (x<<12);
    x = (x^0xc761c23c) ^ (x>>19);
    x = (x+0x165667b1) + (x<<5);
    x = (x+0xd3a2646c) ^ (x<<9);
    x = (x+0xfd7046c5) + (x<<3);
    x = (x^0xb55a4f09) ^ (x>>16);
    return x;
  }
};


template<typename Vector>
void random_fill(Vector &vec)
{
  thrust::tabulate(vec.begin(), vec.end(), hash<typename Vector::value_type>());
}


template<typename T>
void compare(size_t n)
{
  thrust::device_vector<T> unsorted_keys(n), unsorted_values(n), sorted_keys(n), sorted_values(n);

  random_fill(unsorted_keys);
  random_fill(unsorted_values);

  my_sort_by_key(&unsorted_keys, &unsorted_values, &sorted_keys, &sorted_values);
  double my_msecs = time_invocation_cuda(20, my_sort_by_key<T>, &unsorted_keys, &unsorted_values, &sorted_keys, &sorted_values);

  sean_sort_by_key(&unsorted_keys, &unsorted_values, &sorted_keys, &sorted_values);
  double sean_msecs = time_invocation_cuda(20, sean_sort_by_key<T>, &unsorted_keys, &unsorted_values, &sorted_keys, &sorted_values);

  thrust_sort_by_key(&unsorted_keys, &unsorted_values, &sorted_keys, &sorted_values);
  double thrust_msecs = time_invocation_cuda(20, thrust_sort_by_key<T>, &unsorted_keys, &unsorted_values, &sorted_keys, &sorted_values);

  std::cout << "Sean's time: " << sean_msecs << " ms" << std::endl;
  std::cout << "Thrust's time: " << thrust_msecs << " ms" << std::endl;
  std::cout << "My time:       " << my_msecs << " ms" << std::endl;

  std::cout << "Performance relative to Sean: " << sean_msecs / my_msecs << std::endl;
  std::cout << "Performance relative to Thrust: " << thrust_msecs / my_msecs << std::endl;
}


template<typename T>
void validate(size_t n)
{
  thrust::device_vector<T> unsorted_keys(n), unsorted_values(n);

  random_fill(unsorted_keys);
  random_fill(unsorted_values);

  thrust::device_vector<T> ref_keys = unsorted_keys;
  thrust::device_vector<T> ref_values = unsorted_values;
  thrust::sort_by_key(ref_keys.begin(), ref_keys.end(), ref_values.begin());

  thrust::device_vector<T> sorted_keys = unsorted_keys;
  thrust::device_vector<T> sorted_values = unsorted_values;

  my_sort_by_key_(sorted_keys.begin(), sorted_keys.end(), sorted_values.begin(), thrust::less<T>());

  std::cout << "CUDA error: " << hipGetErrorString(hipDeviceSynchronize()) << std::endl;

  assert(sorted_keys == ref_keys);
  assert(sorted_values == ref_values);
}


int main()
{
  size_t n = 12345678;

  //validate<int>(n);
  validate<double>(n);

  std::cout << "Large input: " << std::endl;
  std::cout << "int: " << std::endl;
  compare<int>(n);

  std::cout << "float: " << std::endl;
  compare<float>(n);

  std::cout << "double: " << std::endl;
  compare<double>(n);
  std::cout << std::endl;

  return 0;
}

