#include "hip/hip_runtime.h"
#include <iostream>
#include <moderngpu.cuh>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/detail/minmax.h>
#include <bulk/bulk.hpp>
#include "time_invocation_cuda.hpp"
#include "join_iterator.hpp"


struct stable_sort_each_kernel
{
  template<std::size_t groupsize, std::size_t grainsize, typename RandomAccessIterator1, typename RandomAccessIterator2, typename Compare>
  __device__ void operator()(bulk::concurrent_group<bulk::agent<grainsize>, groupsize> &g, RandomAccessIterator1 keys_first, RandomAccessIterator2 values_first, int count, Compare comp)
  {
    typedef typename bulk::concurrent_group<bulk::agent<grainsize>,groupsize>::size_type size_type;
    const size_type tilesize = groupsize * grainsize;
  
    size_type gid = tilesize * g.index();
    size_type count2 = thrust::min<size_type>(tilesize, count - gid);
  
    bulk::stable_sort_by_key(bulk::bound<tilesize>(g), keys_first + gid, keys_first + gid + count2, values_first + gid, comp);
  }
};


template<typename Size>
__device__
thrust::tuple<Size,Size,Size,Size>
  locate_merge_partitions(Size n, Size group_idx, Size num_groups_per_merge, Size num_elements_per_group, Size mp, Size right_mp)
{
  Size first_group_in_partition = ~(num_groups_per_merge - 1) & group_idx;
  Size partition_size = num_elements_per_group * (num_groups_per_merge >> 1);

  Size partition_first1 = num_elements_per_group * first_group_in_partition;
  Size partition_first2 = partition_first1 + partition_size;

  // Locate diag from the start of the A sublist.
  Size diag = num_elements_per_group * group_idx - partition_first1;
  Size start1 = partition_first1 + mp;
  Size end1 = thrust::min<Size>(n, partition_first1 + right_mp);
  Size start2 = thrust::min<Size>(n, partition_first2 + diag - mp);
  Size end2 = thrust::min<Size>(n, partition_first2 + diag + num_elements_per_group - right_mp);
  
  // The end partition of the last group for each merge operation is computed
  // and stored as the begin partition for the subsequent merge. i.e. it is
  // the same partition but in the wrong coordinate system, so its 0 when it
  // should be listSize. Correct that by checking if this is the last group
  // in this merge operation.
  if(num_groups_per_merge - 1 == ((num_groups_per_merge - 1) & group_idx))
  {
    end1 = thrust::min<Size>(n, partition_first1 + partition_size);
    end2 = thrust::min<Size>(n, partition_first2 + partition_size);
  }

  return thrust::make_tuple(start1, end1, start2, end2);
}


struct merge_by_key_kernel
{
  template<std::size_t groupsize,
           std::size_t grainsize,
           typename RandomAccessIterator1, 
	   typename RandomAccessIterator2,
           typename RandomAccessIterator3,
	   typename RandomAccessIterator4,
	   typename RandomAccessIterator5,
           typename Compare>
  __device__ void operator()(bulk::concurrent_group<bulk::agent<grainsize>, groupsize> &g, RandomAccessIterator1 keys_first, RandomAccessIterator2 values_first, int n, RandomAccessIterator3 merge_paths, int num_groups_per_merge, RandomAccessIterator4 keys_result, RandomAccessIterator5 values_result, Compare comp)
  {
    typedef typename bulk::concurrent_group<bulk::agent<grainsize>, groupsize>::size_type size_type;

    size_type a0, a1, b0, b1;
    thrust::tie(a0, a1, b0, b1) = locate_merge_partitions<size_type>(n, g.index(), num_groups_per_merge, groupsize * grainsize, merge_paths[g.index()], merge_paths[g.index()+1]);
    
    bulk::merge_by_key(bulk::bound<groupsize*grainsize>(g),
                       keys_first + a0, keys_first + a1,
                       keys_first + b0, keys_first + b1,
                       values_first + a0,
                       values_first + b0,
                       keys_result   + groupsize * grainsize * g.index(),
                       values_result + groupsize * grainsize * g.index(),
                       comp);
  }
};


template<typename Iterator, typename Size, typename Compare>
struct locate_merge_path
{
  Iterator haystack_first;
  Size haystack_size;
  Size num_elements_per_group;
  Size num_groups_per_merge;
  thrust::detail::wrapped_function<Compare,bool> comp;

  locate_merge_path(Iterator haystack_first, Size haystack_size, Size num_elements_per_group, Size num_groups_per_merge, Compare comp)
    : haystack_first(haystack_first),
      haystack_size(haystack_size),
      num_elements_per_group(num_elements_per_group),
      num_groups_per_merge(num_groups_per_merge),
      comp(comp)
  {}

  template<typename Index>
  __host__ __device__
  Index operator()(Index merge_path_idx)
  {
    // find the index of the first group that will participate in the eventual merge
    Size first_group_in_partition = ~(num_groups_per_merge - 1) & merge_path_idx;

    // the size of each group's input
    Size size = num_elements_per_group * (num_groups_per_merge / 2);

    // find pointers to the two input arrays
    Size start1 = num_elements_per_group * first_group_in_partition;
    Size start2 = thrust::min<Size>(haystack_size, start1 + size);

    // the size of each input array
    // note we clamp to the end of the total input to handle the last partial list
    Size n1 = thrust::min<Size>(size, haystack_size - start1);
    Size n2 = thrust::min<Size>(size, haystack_size - start2);
    
    // note that diag is computed as an offset from the beginning of the first list
    Size diag = thrust::min<Size>(n1 + n2, num_elements_per_group * merge_path_idx - start1);

    return bulk::merge_path(haystack_first + start1, n1, haystack_first + start2, n2, diag, comp);
  }
};


template<typename DerivedPolicy, typename Iterator1, typename Size1, typename Iterator2, typename Size2, typename Compare>
void locate_merge_paths_(thrust::system::cuda::execution_policy<DerivedPolicy> &exec,
                         Iterator1 result,
                         Size1 n,
                         Iterator2 haystack_first,
                         Size2 haystack_size,
                         Size2 num_elements_per_group,
                         Size2 num_groups_per_merge,
                         Compare comp)
{
  locate_merge_path<Iterator2,Size2,Compare> f(haystack_first, haystack_size, num_elements_per_group, num_groups_per_merge, comp);

  thrust::tabulate(exec, result, result + n, f);
}


template<typename KeyType, typename ValType, typename Comp>
void MergesortPairs(KeyType* keys_global, ValType* values_global, int n, Comp comp, mgpu::CudaContext& context)
{
  typedef int size_type;

  const size_type groupsize = 256;
  const size_type grainsize = 11;
  
  const size_type tilesize = groupsize * grainsize;
  size_type num_groups = (n + tilesize - 1) / tilesize;
  size_type num_passes = mgpu::FindLog2(num_groups, true);
  
  MGPU_MEM(KeyType) keysDestDevice = context.Malloc<KeyType>(n);
  MGPU_MEM(ValType) valsDestDevice = context.Malloc<ValType>(n);

  KeyType* keysSource = keys_global;
  KeyType* keysDest = keysDestDevice->get();
  ValType* valsSource = values_global;
  ValType* valsDest = valsDestDevice->get();

  size_type heap_size = tilesize * thrust::max(sizeof(KeyType), sizeof(ValType));

  bulk::async(bulk::grid<groupsize,grainsize>(num_groups, heap_size), stable_sort_each_kernel(), bulk::root.this_exec, keysSource, valsSource, n, comp);

  // XXX forward exec from parameters here
  thrust::cuda::tag exec;
  thrust::detail::temporary_array<size_type,thrust::cuda::tag> merge_paths(exec, num_groups + 1);
  
  for(size_type pass = 0; pass < num_passes; ++pass) 
  {
    size_type num_groups_per_merge = 2 << pass;

    locate_merge_paths_(exec, merge_paths.begin(), merge_paths.size(), keysSource, n, tilesize, num_groups_per_merge, comp);
    
    size_type heap_size = tilesize * thrust::max(sizeof(KeyType), sizeof(size_type));
    bulk::async(bulk::grid<groupsize,grainsize>(num_groups, heap_size), merge_by_key_kernel(), bulk::root.this_exec, keysSource, valsSource, n, merge_paths.begin(), num_groups_per_merge, keysDest, valsDest, comp);

    std::swap(keysDest, keysSource);
    std::swap(valsDest, valsSource);
  }

  if(1 & num_passes)
  {
    thrust::copy_n(thrust::cuda::tag(), thrust::make_zip_iterator(thrust::make_tuple(keysSource, valsSource)), n, thrust::make_zip_iterator(thrust::make_tuple(keysDest, valsDest)));
  }
}


template<typename RandomAccessIterator1, typename RandomAccessIterator2, typename Compare>
void my_sort_by_key_(RandomAccessIterator1 keys_first, RandomAccessIterator1 keys_last, RandomAccessIterator2 values_first, Compare comp)
{
  mgpu::ContextPtr ctx = mgpu::CreateCudaDevice(0);
  ::MergesortPairs(thrust::raw_pointer_cast(&*keys_first),
                   thrust::raw_pointer_cast(&*values_first),
                   keys_last - keys_first,
                   comp,
                   *ctx);
}


struct my_less
{
  template<typename T>
  __host__ __device__
  bool operator()(const T &x, const T& y)
  {
    return x < y;
  }
};


template<typename T>
void my_sort_by_key(const thrust::device_vector<T> *unsorted_keys,
                    const thrust::device_vector<T> *unsorted_values,
                    thrust::device_vector<T> *sorted_keys,
                    thrust::device_vector<T> *sorted_values)
{
  *sorted_keys = *unsorted_keys;
  *sorted_values = *unsorted_values;
  my_sort_by_key_(sorted_keys->begin(), sorted_keys->end(), sorted_values->begin(), my_less());
}


template<typename T>
void sean_sort_by_key(const thrust::device_vector<T> *unsorted_keys,
                      const thrust::device_vector<T> *unsorted_values,                    
                      thrust::device_vector<T> *sorted_keys,
                      thrust::device_vector<T> *sorted_values)
{
  *sorted_keys = *unsorted_keys;
  *sorted_values = *unsorted_values;
  mgpu::ContextPtr ctx = mgpu::CreateCudaDevice(0);
  mgpu::MergesortPairs(thrust::raw_pointer_cast(sorted_keys->data()),
                       thrust::raw_pointer_cast(sorted_values->data()),
                       sorted_keys->size(),
                       my_less(),
                       *ctx);
}


template<typename T>
void thrust_sort_by_key(const thrust::device_vector<T> *unsorted_keys,
                        const thrust::device_vector<T> *unsorted_values,
                        thrust::device_vector<T> *sorted_keys,
                        thrust::device_vector<T> *sorted_values)
{
  *sorted_keys = *unsorted_keys;
  *sorted_values = *unsorted_values;
  thrust::sort_by_key(sorted_keys->begin(), sorted_keys->end(), sorted_values->begin(), my_less());
}


template<typename T>
struct hash
{
  template<typename Integer>
  __device__ __device__
  T operator()(Integer x)
  {
    x = (x+0x7ed55d16) + (x<<12);
    x = (x^0xc761c23c) ^ (x>>19);
    x = (x+0x165667b1) + (x<<5);
    x = (x+0xd3a2646c) ^ (x<<9);
    x = (x+0xfd7046c5) + (x<<3);
    x = (x^0xb55a4f09) ^ (x>>16);
    return x;
  }
};


template<typename Vector>
void random_fill(Vector &vec)
{
  thrust::tabulate(vec.begin(), vec.end(), hash<typename Vector::value_type>());
}


template<typename T>
void compare(size_t n)
{
  thrust::device_vector<T> unsorted_keys(n), unsorted_values(n), sorted_keys(n), sorted_values(n);

  random_fill(unsorted_keys);
  random_fill(unsorted_values);

  my_sort_by_key(&unsorted_keys, &unsorted_values, &sorted_keys, &sorted_values);
  double my_msecs = time_invocation_cuda(20, my_sort_by_key<T>, &unsorted_keys, &unsorted_values, &sorted_keys, &sorted_values);

  sean_sort_by_key(&unsorted_keys, &unsorted_values, &sorted_keys, &sorted_values);
  double sean_msecs = time_invocation_cuda(20, sean_sort_by_key<T>, &unsorted_keys, &unsorted_values, &sorted_keys, &sorted_values);

  thrust_sort_by_key(&unsorted_keys, &unsorted_values, &sorted_keys, &sorted_values);
  double thrust_msecs = time_invocation_cuda(20, thrust_sort_by_key<T>, &unsorted_keys, &unsorted_values, &sorted_keys, &sorted_values);

  std::cout << "Sean's time: " << sean_msecs << " ms" << std::endl;
  std::cout << "Thrust's time: " << thrust_msecs << " ms" << std::endl;
  std::cout << "My time:       " << my_msecs << " ms" << std::endl;

  std::cout << "Performance relative to Sean: " << sean_msecs / my_msecs << std::endl;
  std::cout << "Performance relative to Thrust: " << thrust_msecs / my_msecs << std::endl;
}


template<typename T>
void validate(size_t n)
{
  thrust::device_vector<T> unsorted_keys(n), unsorted_values(n);

  random_fill(unsorted_keys);
  random_fill(unsorted_values);

  thrust::device_vector<T> ref_keys = unsorted_keys;
  thrust::device_vector<T> ref_values = unsorted_values;
  thrust::sort_by_key(ref_keys.begin(), ref_keys.end(), ref_values.begin(), my_less());

  thrust::device_vector<T> sorted_keys = unsorted_keys;
  thrust::device_vector<T> sorted_values = unsorted_values;

  my_sort_by_key_(sorted_keys.begin(), sorted_keys.end(), sorted_values.begin(), my_less());

  std::cout << "CUDA error: " << hipGetErrorString(hipDeviceSynchronize()) << std::endl;

  if(n < 30 && sorted_keys != ref_keys)
  {
    std::cerr << "reference: " << std::endl;

    for(int i = 0; i < n; ++i)
    {
      std::cerr << ref_keys[i] << " ";
    }

    std::cerr << std::endl;


    std::cerr << "output: " << std::endl;

    for(int i = 0; i < n; ++i)
    {
      std::cerr << sorted_keys[i] << " ";
    }

    std::cerr << std::endl;
  }

  assert(sorted_keys == ref_keys);
  assert(sorted_values == ref_values);
}


int main()
{
  std::cout << "small input: " << std::endl;
  std::cout << "int: " << std::endl;

  validate<int>(20);

  size_t n = 12345678;

  std::cout << "Large input: " << std::endl;
  std::cout << "int: " << std::endl;
  compare<int>(n);

  std::cout << "float: " << std::endl;
  compare<float>(n);

  std::cout << "double: " << std::endl;
  compare<double>(n);
  std::cout << std::endl;

  return 0;
}

